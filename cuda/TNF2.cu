#include "hip/hip_runtime.h"
// nvcc TNF.cu -lz
// ta bien
#include "../extra/KseqReader.h"
#include <algorithm>
#include <chrono>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <thread>
#include <unordered_map>
#include <unordered_set>
#include <vector>

__device__ __constant__ int n_TNF_d = 136;

__device__ __constant__ unsigned char TNmap_d[256] = {
    2,   21,  31,  115, 101, 119, 67,  50,  135, 126, 69,  92,  116, 88,  8,   78,  47,  96,  3,   70,  106, 38,
    48,  83,  16,  22,  136, 114, 5,   54,  107, 120, 72,  41,  44,  26,  27,  23,  136, 53,  12,  81,  136, 127,
    30,  110, 136, 80,  132, 123, 71,  102, 79,  1,   35,  124, 29,  4,   136, 34,  91,  17,  136, 52,  9,   77,
    136, 117, 76,  93,  136, 65,  6,   73,  136, 68,  28,  94,  136, 113, 121, 36,  136, 10,  103, 99,  136, 87,
    129, 14,  136, 136, 98,  19,  136, 97,  15,  56,  136, 131, 57,  46,  136, 136, 122, 60,  136, 136, 42,  62,
    136, 136, 7,   130, 136, 51,  133, 20,  136, 134, 89,  86,  136, 136, 104, 95,  136, 136, 49,  136, 136, 136,
    105, 136, 136, 136, 33,  136, 136, 136, 43,  136, 136, 136, 55,  136, 136, 136, 112, 136, 136, 136, 136, 136,
    136, 136, 75,  136, 136, 136, 32,  136, 136, 136, 136, 136, 136, 136, 136, 136, 136, 136, 136, 136, 136, 136,
    100, 136, 136, 136, 63,  136, 136, 136, 136, 136, 136, 136, 136, 136, 136, 136, 125, 108, 136, 136, 58,  24,
    136, 136, 84,  13,  136, 136, 25,  66,  136, 136, 18,  128, 136, 136, 74,  61,  136, 136, 85,  136, 136, 136,
    118, 40,  136, 136, 109, 90,  136, 136, 45,  136, 136, 136, 111, 136, 136, 136, 82,  136, 136, 136, 59,  11,
    136, 136, 64,  37,  136, 136, 0,   136, 136, 136, 39,  136, 136, 136};
__device__ __constant__ unsigned char TNPmap_d[256] = {
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1,
    0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

__device__ const char *get_contig_d(int contig_index, const char *seqs_d, const size_t *seqs_d_index)
{
    size_t contig_beg = 0;
    if (contig_index != 0)
    {
        contig_beg = seqs_d_index[contig_index - 1];
    }
    return seqs_d + contig_beg;
}

__device__ __host__ unsigned char get_tn(const char *contig, size_t index)
{
    unsigned char tn = 0;
    for (int i = 0; i < 4; i++)
    {
        char N = contig[index + i];
        if (N == 'A')
            N = 0;
        else if (N == 'C')
            N = 1;
        else if (N == 'T')
            N = 2;
        else if (N == 'G')
            N = 3;
        else
            return 170; // no existe en TNmap[]

        tn = (tn << 2) + N;
    }
    return tn;
}

__device__ unsigned char get_revComp_tn_d(const char *contig, size_t index)
{
    unsigned char tn = 0;
    for (int i = 3; i >= 0; i--)
    {
        char N = contig[index + i];
        if (N == 'A')
            N = 2;
        else if (N == 'C')
            N = 3;
        else if (N == 'T')
            N = 0;
        else if (N == 'G')
            N = 1;
        else
            return 170; // no existe en TNmap[]
        tn = (tn << 2) + N;
    }
    return tn;
}

__global__ void get_TNF(double *TNF_d, const char *seqs_d, const size_t *seqs_d_index, size_t nobs,
                        const unsigned char *smallCtgs, size_t contigs_per_thread)
{
    size_t thead_id = threadIdx.x + blockIdx.x * blockDim.x;

    for (size_t i = 0; i < contigs_per_thread; i++)
    {
        size_t contig_index = (thead_id * contigs_per_thread) + i;
        if (contig_index >= nobs)
            break;
        for (int j = 0; j < n_TNF_d; j++)
        {
            TNF_d[contig_index * n_TNF_d + j] = 0;
        }
    }

    for (size_t i = 0; i < contigs_per_thread; i++)
    {
        size_t contig_index = (thead_id * contigs_per_thread) + i;
        if (contig_index >= nobs)
            break;
        if (smallCtgs[contig_index] == 0)
        {
            const char *contig = get_contig_d(contig_index, seqs_d, seqs_d_index);
            size_t contig_size = seqs_d_index[contig_index];
            if (contig_index != 0)
            {
                contig_size -= seqs_d_index[contig_index - 1];
            }
            for (size_t j = 0; j < contig_size - 3; ++j)
            {
                unsigned char tn = get_tn(contig, j);
                // SI tn NO SE ENCUENTRA EN TNmap el complemento del palindromo sí estará
                if (TNmap_d[tn] != n_TNF_d)
                {
                    ++TNF_d[contig_index * n_TNF_d + TNmap_d[tn]];
                }

                tn = get_revComp_tn_d(contig, j);

                // SALTA EL PALINDROMO PARA NO INSERTARLO NUEVAMENTE
                if (TNPmap_d[tn] == 0)
                {
                    if (TNmap_d[tn] != n_TNF_d)
                    {
                        ++TNF_d[contig_index * n_TNF_d + TNmap_d[tn]];
                    }
                }
            }
            double rsum = 0;
            for (size_t c = 0; c < n_TNF_d; ++c)
            {
                rsum += TNF_d[contig_index * n_TNF_d + c] * TNF_d[contig_index * n_TNF_d + c];
            }
            rsum = sqrt(rsum);
            for (size_t c = 0; c < n_TNF_d; ++c)
            {
                TNF_d[contig_index * n_TNF_d + c] /= rsum; // OK
            }
        }
    }
}

static const std::string TN[] = {
    "GGTA", "AGCC", "AAAA", "ACAT", "AGTC", "ACGA", "CATA", "CGAA", "AAGT", "CAAA", "CCAG", "GGAC", "ATTA", "GATC",
    "CCTC", "CTAA", "ACTA", "AGGC", "GCAA", "CCGC", "CGCC", "AAAC", "ACTC", "ATCC", "GACC", "GAGA", "ATAG", "ATCA",
    "CAGA", "AGTA", "ATGA", "AAAT", "TTAA", "TATA", "AGTG", "AGCT", "CCAC", "GGCC", "ACCC", "GGGA", "GCGC", "ATAC",
    "CTGA", "TAGA", "ATAT", "GTCA", "CTCC", "ACAA", "ACCT", "TAAA", "AACG", "CGAG", "AGGG", "ATCG", "ACGC", "TCAA",
    "CTAC", "CTCA", "GACA", "GGAA", "CTTC", "GCCC", "CTGC", "TGCA", "GGCA", "CACG", "GAGC", "AACT", "CATG", "AATT",
    "ACAG", "AGAT", "ATAA", "CATC", "GCCA", "TCGA", "CACA", "CAAC", "AAGG", "AGCA", "ATGG", "ATTC", "GTGA", "ACCG",
    "GATA", "GCTA", "CGTC", "CCCG", "AAGC", "CGTA", "GTAC", "AGGA", "AATG", "CACC", "CAGC", "CGGC", "ACAC", "CCGG",
    "CCGA", "CCCC", "TGAA", "AACA", "AGAG", "CCCA", "CGGA", "TACA", "ACCA", "ACGT", "GAAC", "GTAA", "ATGC", "GTTA",
    "TCCA", "CAGG", "ACTG", "AAAG", "AAGA", "CAAG", "GCGA", "AACC", "ACGG", "CCAA", "CTTA", "AGAC", "AGCG", "GAAA",
    "AATC", "ATTG", "GCAC", "CCTA", "CGAC", "CTAG", "AGAA", "CGCA", "CGCG", "AATA"};

static const std::string TNP[] = {"ACGT", "AGCT", "TCGA", "TGCA", "CATG", "CTAG", "GATC", "GTAC",
                                  "ATAT", "TATA", "CGCG", "GCGC", "AATT", "TTAA", "CCGG", "GGCC"};
int n_STREAMS = 1;
int n_THREADS = 32;
int n_BLOCKS = 128;

std::vector<std::string> seqs;
std::unordered_map<size_t, size_t> gCtgIdx;
std::unordered_set<int> smallCtgs;

const int n_TNF = 136;
const int n_TNFP = 16;

unsigned char TNmap[256];
unsigned char TNPmap[256];

static size_t minContig = 2500;               // minimum contig size for binning
static size_t minContigByCorr = 1000;         // minimum contig size for recruiting (by abundance correlation)
static size_t minContigByCorrForGraph = 1000; // for graph generation purpose

std::thread *streams;
double **TNF_d;
char **seqs_d;
size_t **seqs_d_index;
unsigned char **smallCtgs_d;

size_t nobs_cont;
size_t kernel_cont;
std::string *seqs_kernel;
std::vector<double *> TNF;
size_t *seqs_kernel_index;
unsigned char *smallCtgs_kernel;

void kernel(dim3 blkDim, dim3 grdDim, int cont)
{
    hipStream_t stream;
    hipStreamCreate(&stream);

    int index = cont % n_STREAMS;

    // std::cout << "kernel: " << kernel_cont<< std::endl;
    hipMalloc(&seqs_d[index], seqs_kernel[index].size());
    hipMemcpy(seqs_d[index], seqs_kernel[index].data(), seqs_kernel[index].size(), hipMemcpyHostToDevice);
    hipMemcpy(seqs_d_index[index], &seqs_kernel_index[index * n_THREADS * n_BLOCKS * sizeof(size_t)],
               n_BLOCKS * n_THREADS * sizeof(size_t), hipMemcpyHostToDevice); // seqs_index
    hipMemcpy(smallCtgs_d[index], &smallCtgs_kernel[index * n_THREADS * n_BLOCKS], n_BLOCKS * n_THREADS,
               hipMemcpyHostToDevice);

    get_TNF<<<grdDim, blkDim, 0, stream>>>(TNF_d[index], seqs_d[index], seqs_d_index[index], nobs_cont,
                                           smallCtgs_d[index], 1);

    hipStreamSynchronize(stream);
    hipFree(seqs_d[index]);
    hipMemcpy(TNF[cont], TNF_d[index], n_BLOCKS * n_THREADS * n_TNF * sizeof(double), hipMemcpyDeviceToHost);

    hipStreamDestroy(stream);

    seqs_kernel[index] = "";
}

int main(int argc, char const *argv[])
{
    if (argc > 3)
    {
        n_BLOCKS = atoi(argv[1]);
        n_THREADS = atoi(argv[2]);
        n_STREAMS = atoi(argv[3]);
    }
    // std::cout << "n°bloques: "<< n_BLOCKS <<", n°threads:"<< n_THREADS << std::endl;

    // se inicializan los mapas
    for (int i = 0; i < 256; i++)
    {
        TNmap[i] = n_TNF;
        TNPmap[i] = 0;
    }
    for (int i = 0; i < n_TNF; ++i)
    {
        unsigned char key = get_tn(TN[i].c_str(), 0);
        TNmap[key] = i;
    }

    for (size_t i = 0; i < n_TNFP; ++i)
    {
        unsigned char key = get_tn(TNP[i].c_str(), 0);
        TNPmap[key] = 1;
    }

    auto start_global = std::chrono::system_clock::now();
    auto start = std::chrono::system_clock::now();

    // crear streams
    streams = new std::thread[n_STREAMS];

    seqs_kernel = new std::string[n_STREAMS];

    TNF_d = new double *[n_STREAMS];
    seqs_d = new char *[n_STREAMS];
    seqs_d_index = new size_t *[n_STREAMS];
    smallCtgs_d = new unsigned char *[n_STREAMS];

    bool bool_thread[n_STREAMS];
    dim3 blkDim(n_THREADS, 1, 1);
    dim3 grdDim(n_BLOCKS, 1, 1);

    nobs_cont = 0;
    kernel_cont = 0;
    seqs_kernel_index = (size_t *)malloc(n_THREADS * n_BLOCKS * sizeof(size_t) * n_STREAMS);
    smallCtgs_kernel = (unsigned char *)malloc(n_THREADS * n_BLOCKS * n_STREAMS);

    for (int i = 0; i < n_STREAMS; i++)
    {
        seqs_kernel[i] = "";
        bool_thread[i] = 0;
        hipMalloc(&TNF_d[i], n_BLOCKS * n_THREADS * n_TNF * sizeof(double));
        hipMalloc(&seqs_d_index[i], n_BLOCKS * n_THREADS * sizeof(size_t));
        hipMalloc(&smallCtgs_d[i], n_BLOCKS * n_THREADS);
    }

    size_t nobs = 0;

    int nresv = 0;
    std::string inFile = "test.gz";

    gzFile f = gzopen(inFile.c_str(), "r");
    if (f == NULL)
    {
        cerr << "[Error!] can't open the sequence fasta file " << inFile << endl;
        return 1;
    }
    else
    {
        kseq_t *kseq = kseq_init(f);
        int64_t len;
        while ((len = kseq_read(kseq)) > 0)
        {
            std::transform(kseq->seq.s, kseq->seq.s + len, kseq->seq.s, ::toupper);
            if (kseq->name.l > 0)
            {
                std::cout <<"."; 
                size_t index = (kernel_cont % n_STREAMS) * n_THREADS * n_BLOCKS;
                if (bool_thread[kernel_cont % n_STREAMS] && kernel_cont % n_STREAMS < kernel_cont)
                {
                    bool_thread[kernel_cont % n_STREAMS] = 0;
                    streams[kernel_cont % n_STREAMS].join();
                }
                std::cout <<","; 

                if (len >= (int)std::min(minContigByCorr, minContigByCorrForGraph))
                {
                    if (len < (int)minContig)
                    {
                        if (len >= (int)minContigByCorr)
                        {
                            smallCtgs.insert(1);
                            smallCtgs_kernel[index + nobs_cont] = 1;
                        }
                        else
                        {
                            smallCtgs_kernel[index + nobs_cont] = 0;
                            ++nresv;
                        }
                    }
                    else
                    {
                        smallCtgs_kernel[index + nobs_cont] = 0;
                    }
                    gCtgIdx[nobs++] = seqs.size();

                    seqs_kernel[kernel_cont % n_STREAMS] += kseq->seq.s;
                    seqs_kernel_index[index * sizeof(size_t) + nobs_cont] = seqs_kernel[kernel_cont % n_STREAMS].size();
                    nobs_cont++;
                }
                else
                {
                    // ignored[kseq->name.s] = seqs.size();
                }
                // contig_names.push_back(kseq->name.s);
                seqs.push_back(kseq->seq.s);

                if (nobs_cont == n_BLOCKS * n_THREADS)
                {
                    std::cout << "hola:"<< kernel_cont % n_STREAMS<< std::endl;
                    if (bool_thread[kernel_cont % n_STREAMS] && kernel_cont % n_STREAMS < kernel_cont){
                        bool_thread[kernel_cont % n_STREAMS] = 0;
                        streams[kernel_cont % n_STREAMS].join();
                    }   
                    TNF.emplace_back((double *)malloc(n_BLOCKS * n_THREADS * n_TNF * sizeof(double)));
                    streams[kernel_cont % n_STREAMS] = std::thread(kernel, blkDim, grdDim, kernel_cont);
                    std::cout << "adios"<< kernel_cont % n_STREAMS<< std::endl;
                    bool_thread[kernel_cont % n_STREAMS] = 1;
                    kernel_cont++;
                    nobs_cont = 0;
                }
            }
        }
        kseq_destroy(kseq);
        kseq = NULL;
        gzclose(f);
    }
    if (nobs_cont != 0)
    {
        if (bool_thread[kernel_cont % n_STREAMS] && kernel_cont % n_STREAMS < kernel_cont)
        {
            streams[kernel_cont % n_STREAMS].join();
        }
        TNF.emplace_back((double *)malloc(n_BLOCKS * n_THREADS * n_TNF * sizeof(double)));
        streams[kernel_cont % n_STREAMS] = std::thread(kernel, blkDim, grdDim, kernel_cont);
        kernel_cont++;
        nobs_cont = 0;
    }

    for (int i = 0; i < std::min((size_t)n_STREAMS, kernel_cont); i++)
    {
        bool_thread[i] = 0;
        streams[i].join();
    }

    auto end = std::chrono::system_clock::now();
    std::chrono::duration<float, std::milli> duration = end - start;
    // std::cout <<"leer contigs + procesamiento "<< duration.count()/1000.f << "s " << std::endl;

    auto end_global = std::chrono::system_clock::now();
    duration = end_global - start_global;
    std::cout << duration.count() / 1000.f << std::endl;

    std::ofstream out("TNF.bin", ios::out | ios::binary);
    if (out)
    {
        for (size_t i = 0; i < TNF.size(); i++)
        {
            if (i < (TNF.size() - 1) || nobs % (n_BLOCKS * n_THREADS) == 0)
                out.write((char *)TNF[i], n_BLOCKS * n_THREADS * n_TNF * sizeof(double));
            else
                out.write((char *)TNF[i], (nobs % (n_BLOCKS * n_THREADS)) * n_TNF * sizeof(double));
        }
        // std::cout << "TNF guardado" << std::endl;
    }
    else
    {
        // std::cout << "Error al guardar" << std::endl;
    }
    out.close();

    for (int i = 0; i < TNF.size(); i++)
    {
        free(TNF[i]);
    }
    return 0;
}
