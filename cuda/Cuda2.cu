#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <immintrin.h>
#include <omp.h>
#include <sys/stat.h>
#include <sys/time.h>
#include <sys/types.h>

#include <algorithm>
#include <chrono>
#include <cstdarg>
#include <fstream>
#include <functional>
#include <iostream>
#include <iterator>
#include <queue>
#include <random>
#include <sstream>
#include <string>
#include <thread>
#include <unordered_map>
#include <unordered_set>
#include <utility>
#include <vector>
#ifdef __APPLE__
#include <mach/mach.h>
#include <sys/sysctl.h>
#else
#include <sys/sysinfo.h>
#endif

#include "../extra/metrictime2.hpp"
#include "ProgressTracker.h"
#include "cuckoohash_map.hh"
#include "ranker.h"
#include "tile.h"

// force BOOST ublas optimizations
#define BOOST_UBLAS_INLINE inline
#define BOOST_UBLAS_CHECK_ENABLE 0
#define BOOST_UBLAS_USE_FAST_SAME
#define BOOST_UBLAS_TYPE_CHECK 0

#include <boost/dynamic_bitset.hpp>
#include <boost/filesystem.hpp>
#include <boost/math/distributions.hpp>
#include <boost/program_options.hpp>
#include <boost/system/error_code.hpp>
#include <string>

#if (BOOST_VERSION / 100000 == 1) && (BOOST_VERSION / 100 % 1000 == 64)
#include <boost/serialization/array_wrapper.hpp>
#endif

#include <boost/numeric/ublas/matrix.hpp>
#include <boost/numeric/ublas/matrix_proxy.hpp>

using std::cerr;
using std::cout;
using std::endl;
namespace po = boost::program_options;

typedef double Distance;
typedef double Similarity;
#define LOG log
#define LOG10 log10
#define SQRT sqrt
#define EXP exp
#define POW pow
#define FABS fabs

typedef boost::math::normal_distribution<Distance> Normal;

static std::string version = "Metabat 2 cuda 0.1";
static std::string DATE = "2023-09-25";
static bool verbose = false;
static bool debug = false;
static bool noBinOut = false;
static size_t minClsSize = 200000;
static size_t minContig = 2500;  // minimum contig size for binning
static std::string inFile;
static std::string abdFile;
static bool cvExt;
static std::string outFile;
static bool onlyLabel = false;
static bool noAdd = false;
static size_t numThreads = 0;
static Similarity maxP = 95;
static Similarity minS = 60;
static Similarity pTNF = 0;
static Distance minCV = 1;
static Distance minCVSum = 1;
static bool saveCls = false;
static bool outUnbinned = false;
static size_t minSample = 3;
static unsigned long long totalSize = 0, totalSize1 = 0;
static int numThreads2 = 32;

static size_t maxEdges = 200;
static const char line_delim = '\n';
static const char tab_delim = '\t';
static const char fasta_delim = '>';
static const std::size_t buf_size = 1024 * 1024;

static char* _mem;
static size_t fsize = 0;
static std::vector<std::string_view> contig_names;
static std::vector<std::string_view> small_contig_names;
static std::vector<std::string_view> seqs;
static std::vector<std::string_view> small_seqs;
static std::vector<Distance> logSizes;
static std::vector<size_t> sizes;
static std::vector<size_t> small_sizes;

typedef std::vector<int> ContigVector;
typedef std::unordered_set<int> ContigSet;
typedef std::unordered_map<int, ContigVector> ClassMap;

static size_t nobs = 0;   // # of large
static size_t nobs1 = 0;  // # of small

static boost::numeric::ublas::matrix<float> ABD;
static boost::numeric::ublas::matrix<float> ABD_VAR;
static boost::numeric::ublas::matrix<float> small_ABD;
// static boost::numeric::ublas::matrix<float> TNF;

// static float* TNF_data;
static float* TNF_d;
static char* seqs_d;
static size_t* seqs_d_index;
static double* contig_log;
std::vector<size_t> seqs_h_index_i;
std::vector<size_t> seqs_h_index_e;

typedef boost::numeric::ublas::matrix_row<boost::numeric::ublas::matrix<float>> MatrixRowType;
typedef boost::numeric::ublas::matrix_column<boost::numeric::ublas::matrix<float>> MatrixColumnType;

static size_t nABD = 0;
static const size_t nTNF = 136;
static unsigned long long seed = 0;

static std::chrono::steady_clock::time_point t1, t2;

static std::vector<int> TNLookup;  // lookup table 0 - 255 of raw 4-mer to tetramer index in TNF

__device__ __constant__ unsigned char TNmap_d[256] = {
    2,   21,  31,  115, 101, 119, 67,  50, 135, 126, 69,  92,  116, 88,  8,   78,  47,  96,  3,   70,  106, 38,  48,  83,  16,  22,
    8,   114, 5,   54,  107, 120, 72,  41, 44,  26,  27,  23,  71,  53,  12,  81,  31,  127, 30,  110, 3,   80,  132, 123, 71,  102,
    79,  1,   35,  124, 29,  4,   67,  34, 91,  17,  48,  52,  9,   77,  127, 117, 76,  93,  34,  65,  6,   73,  92,  68,  28,  94,
    114, 113, 121, 36,  80,  10,  103, 99, 52,  87,  129, 14,  78,  113, 98,  19,  120, 97,  15,  56,  26,  131, 57,  46,  102, 51,
    122, 60,  115, 117, 42,  62,  70,  10, 7,   130, 53,  51,  133, 20,  124, 134, 89,  86,  50,  65,  104, 95,  83,  87,  49,  111,
    12,  122, 105, 0,   29,  89,  33,  84, 135, 6,   43,  85,  16,  129, 55,  82,  30,  42,  112, 39,  91,  104, 43,  25,  116, 28,
    75,  118, 5,   98,  32,  109, 72,  15, 100, 59,  132, 7,   49,  125, 2,   9,   55,  18,  47,  121, 100, 45,  27,  57,  63,  64,
    79,  133, 105, 58,  101, 76,  112, 74, 106, 103, 125, 108, 81,  60,  58,  24,  4,   86,  84,  13,  126, 73,  25,  66,  22,  14,
    18,  128, 110, 62,  74,  61,  17,  95, 85,  66,  88,  94,  118, 40,  54,  19,  109, 90,  41,  56,  45,  11,  123, 130, 111, 108,
    21,  77,  82,  128, 96,  36,  59,  11, 23,  46,  64,  37,  1,   20,  0,   24,  119, 93,  39,  61,  38,  99};

__device__ __constant__ unsigned char BN[256] = {
    4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4,
    4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 0, 4, 1, 4, 4, 4, 3, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 2, 4,
    4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 0, 4, 1, 4, 4, 4, 3, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 2, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4,
    4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4,
    4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4,
    4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4};

__device__ __constant__ double _b1[18] = {46349.1624324381,  -76092.3748553155, -639.918334183,  53873.3933743949, -156.6547554844,
                                          -21263.6010657275, 64.7719132839,     5003.2646455284, -8.5014386744,    -700.5825500292,
                                          0.3968284526,      54.037542743,      -1.7713972342,   474.0850141891,   -23.966597785,
                                          0.7800219061,      -0.0138723693,     0.0001027543};

__device__ __constant__ double _c1[18] = {-443565.465710869, 718862.10804858, 5114.1630934534,   -501588.206183097, 784.4442123743,
                                          194712.394138513,  -377.9645994741, -45088.7863182741, 50.5960513287,     6220.3310639927,
                                          -2.3670776453,     -473.269785487,  15.3213264134,     -3282.8510348085,  164.0438603974,
                                          -5.2778800755,     0.0929379305,    -0.0006826817};

__device__ __constant__ double _b2[17] = {6770.9351457442, -5933.7589419767, -2976.2879986855, 3279.7524685865, 1602.7544794819,
                                          -967.2906583423, -462.0149190219,  159.8317289682,   74.4884405822,   -14.0267151808,
                                          -6.3644917671,   0.5108811613,     0.2252455343,     0.965040193,     -0.0546309127,
                                          0.0012917084,    -1.14383e-05};

__device__ __constant__ double _c2[19] = {39406.5712626297,  -77863.1741143294, 9586.8761567725, 55360.1701572325, -5825.2491611377,
                                          -21887.8400068324, 1751.6803621934,   5158.3764225203, -290.1765894829,  -724.0348081819,
                                          25.364646181,      56.0522105105,     -0.9172073892,   -1.8470088417,    449.4660736502,
                                          -24.4141920625,    0.8465834103,      -0.0158943762,   0.0001235384};
//__device__ __constant__ double floor_preProb = 2.197224577336219564216435173875652253627777099609375;
// más preciso
__device__ __constant__ double floor_preProb = 2.1972245773362193827904904738450514092949811156454989034693886672;

__device__ double cal_tnf_pre_dist_d(double r1, double r2, const float* __restrict__ TNF1, const float* __restrict__ TNF2) {
    double d = 0.0;
    float _diff;
    for (size_t i = 0; i < 136; ++i) {
        _diff = TNF1[i] - TNF2[i];
        d += _diff * _diff;
    }

    d = sqrt(d);

    double b, c;

    double lw[19];
    lw[0] = min(r1, r2);
    lw[1] = max(r1, r2);
    lw[2] = lw[0] * lw[0];
    lw[4] = lw[2] * lw[0];
    lw[6] = lw[4] * lw[0];
    lw[8] = lw[6] * lw[0];
    lw[10] = lw[8] * lw[0];
    lw[11] = lw[10] * lw[0];
    lw[3] = lw[1] * lw[1];
    lw[5] = lw[3] * lw[1];
    lw[7] = lw[5] * lw[1];
    lw[9] = lw[7] * lw[1];
    lw[12] = lw[0] * lw[1];
    lw[14] = lw[4] * lw[5];
    lw[15] = lw[6] * lw[7];
    lw[16] = lw[8] * lw[9];
    lw[13] = lw[2] * lw[3];
    lw[18] = lw[9] * lw[1];

    double4 _b4, _b14, _lw4;

    b = _b1[0];
    for (size_t i = 0; i < 16; i += 4) {
        _b14 = *reinterpret_cast<double*>(_b1 + i + 1);
        _lw4 = *reinterpret_cast<double*>(lw + i);
        _b4 += _b14 * _lw4;
    }
    b += _b4.x + _b4.y + _b4.z + _b4.w;
    b += _b1[17] * lw[16];

    /*
    b = _b1[0] + _b1[1] * lw[0] + _b1[2] * lw[1] + _b1[3] * lw[2] + _b1[4] * lw[3] + _b1[5] * lw[4] + _b1[6] * lw[5] + _b1[7] * lw[6] +
        _b1[8] * lw[7] + _b1[9] * lw[8] + _b1[10] * lw[9] + _b1[11] * lw[10] + _b1[12] * lw[11] + _b1[13] * lw[12] + _b1[14] * lw[13] +
        _b1[15] * lw[14] + _b1[16] * lw[15] + _b1[17] * lw[16];
    */

    c = _c1[0];
    for (size_t i = 0; i < 17; i++) {
        c += _c1[i + 1] * lw[i];
    }
    /*
    c = _c1[0] + _c1[1] * lw[0] + _c1[2] * lw[1] + _c1[3] * lw[2] + _c1[4] * lw[3] + _c1[5] * lw[4] + _c1[6] * lw[5] + _c1[7] * lw[6] +
        _c1[8] * lw[7] + _c1[9] * lw[8] + _c1[10] * lw[9] + _c1[11] * lw[10] + _c1[12] * lw[11] + _c1[13] * lw[12] + _c1[14] * lw[13] +
        _c1[15] * lw[14] + _c1[16] * lw[15] + _c1[17] * lw[16];
    */
    double preProb = -(b + c * d);

    if (preProb <= floor_preProb) {
        b = _b2[0] + _b2[1] * lw[0] + _b2[2] * lw[1] + _b2[3] * lw[2] + _b2[4] * lw[3] + _b2[5] * lw[4] + _b2[6] * lw[5] +
            _b2[7] * lw[6] + _b2[8] * lw[7] + _b2[9] * lw[8] + _b2[10] * lw[9] + _b2[11] * lw[10] + _b2[12] * lw[18] +
            _b2[13] * lw[13] + _b2[14] * lw[14] + _b2[15] * lw[15] + _b2[16] * lw[16];
        c = _c2[0] + _c2[1] * lw[0] + _c2[2] * lw[1] + _c2[3] * lw[2] + _c2[4] * lw[3] + _c2[5] * lw[4] + _c2[6] * lw[5] +
            _c2[7] * lw[6] + _c2[8] * lw[7] + _c2[9] * lw[8] + _c2[10] * lw[9] + _c2[11] * lw[10] + _c2[12] * lw[18] +
            _c2[13] * lw[11] + _c2[14] * lw[12] + _c2[15] * lw[13] + _c2[16] * lw[14] + _c2[17] * lw[15] + _c2[18] * lw[16];
        preProb = -(b + c * d);
        if (preProb > floor_preProb) preProb = floor_preProb;
    }
    return preProb;
}

/*
__device__ double cal_tnf_dist_d(double r1, double r2, float* TNF1, float* TNF2) {
    double d = 0.0;
    float tn1, tn2, _diff;
    for (size_t i = 0; i < 136; ++i) {
        tn1 = TNF1[i];
        tn2 = TNF2[i];
        _diff = tn1 - tn2;
        d += _diff * _diff;
    }

    d = sqrt(d);

    double b, c;

    double lw[19];
    lw[0] = min(r1, r2);
    lw[1] = max(r1, r2);
    lw[2] = lw[0] * lw[0];
    lw[4] = lw[2] * lw[0];
    lw[6] = lw[4] * lw[0];
    lw[8] = lw[6] * lw[0];
    lw[10] = lw[8] * lw[0];
    lw[11] = lw[10] * lw[0];
    lw[3] = lw[1] * lw[1];
    lw[5] = lw[3] * lw[1];
    lw[7] = lw[5] * lw[1];
    lw[9] = lw[7] * lw[1];
    lw[12] = lw[0] * lw[1];
    lw[14] = lw[4] * lw[5];
    lw[15] = lw[6] * lw[7];
    lw[16] = lw[8] * lw[9];
    lw[13] = lw[2] * lw[3];
    lw[18] = lw[9] * lw[1];

    double prob;

    b = _b1[0] + _b1[1] * lw[0] + _b1[2] * lw[1] + _b1[3] * lw[2] + _b1[4] * lw[3] + _b1[5] * lw[4] + _b1[6] * lw[5] + _b1[7] * lw[6] +
        _b1[8] * lw[7] + _b1[9] * lw[8] + _b1[10] * lw[9] + _b1[11] * lw[10] + _b1[12] * lw[11] + _b1[13] * lw[12] + _b1[14] * lw[13] +
        _b1[15] * lw[14] + _b1[16] * lw[15] + _b1[17] * lw[16];

    c = _c1[0] + _c1[1] * lw[0] + _c1[2] * lw[1] + _c1[3] * lw[2] + _c1[4] * lw[3] + _c1[5] * lw[4] + _c1[6] * lw[5] + _c1[7] * lw[6] +
        _c1[8] * lw[7] + _c1[9] * lw[8] + _c1[10] * lw[9] + _c1[11] * lw[10] + _c1[12] * lw[11] + _c1[13] * lw[12] + _c1[14] * lw[13] +
        _c1[15] * lw[14] + _c1[16] * lw[15] + _c1[17] * lw[16];

    double preProb = -(b + c * d);
    prob = preProb <= floor_preProb ? 0.1 : 1.0 / (1 + exp(preProb));

    if (prob == 0.1) {
        b = _b2[0] + _b2[1] * lw[0] + _b2[2] * lw[1] + _b2[3] * lw[2] + _b2[4] * lw[3] + _b2[5] * lw[4] + _b2[6] * lw[5] +
            _b2[7] * lw[6] + _b2[8] * lw[7] + _b2[9] * lw[8] + _b2[10] * lw[9] + _b2[11] * lw[10] + _b2[12] * lw[18] +
            _b2[13] * lw[13] + _b2[14] * lw[14] + _b2[15] * lw[15] + _b2[16] * lw[16];
        c = _c2[0] + _c2[1] * lw[0] + _c2[2] * lw[1] + _c2[3] * lw[2] + _c2[4] * lw[3] + _c2[5] * lw[4] + _c2[6] * lw[5] +
            _c2[7] * lw[6] + _c2[8] * lw[7] + _c2[9] * lw[8] + _c2[10] * lw[9] + _c2[11] * lw[10] + _c2[12] * lw[18] +
            _c2[13] * lw[11] + _c2[14] * lw[12] + _c2[15] * lw[13] + _c2[16] * lw[14] + _c2[17] * lw[15] + _c2[18] * lw[16];
        preProb = -(b + c * d);
        prob = preProb <= floor_preProb ? 1.0 / (1 + exp(preProb)) : 0.1;
    }
    return prob;
}
*/

__global__ void get_tnf_graph(double* graph, const float* __restrict__ TNF, const double* __restrict__ contig_log, size_t nc1,
                              size_t nc2, size_t off1, size_t off2, double floor_preProb_cutoff) {
    size_t prob_index = (threadIdx.x + blockIdx.x * blockDim.x);
    size_t r1 = prob_index / nc2;
    if (r1 >= nc1) return;
    size_t ct1 = off1 + r1;
    size_t ct2 = off2 + prob_index % nc2;
    if (ct1 == ct2) return;
    double preProb = cal_tnf_pre_dist_d(contig_log[ct1], contig_log[ct2], TNF + ct1 * 136, TNF + ct2 * 136);
    if (preProb > floor_preProb_cutoff)
        graph[prob_index] = 1.0 - (1.0 / (1 + exp(preProb)));
    else
        graph[prob_index] = 0.0;
}

__global__ void get_tnf_max_prob_sample3(double* max_dist, const float* __restrict__ TNF, double* size_log, size_t* contigs,
                                         size_t nobs, size_t _des, size_t limit) {
    extern __shared__ double shared_max[];
    size_t contig_idx = _des + blockIdx.x;
    if (contig_idx >= limit) return;
    double local_max = -1;
    float TNF1[136];
    for (int i = 0; i < 136; i++) {
        TNF1[i] = TNF[contigs[contig_idx] * 136 + i];
    }
    size_t dist_per_thread = (nobs + blockDim.x - 1) / blockDim.x;
    for (size_t i = dist_per_thread * threadIdx.x; i < min(dist_per_thread * threadIdx.x + dist_per_thread, nobs); i++) {
        if (i == contig_idx) continue;
        double dist = cal_tnf_pre_dist_d(size_log[contigs[contig_idx]], size_log[contigs[i]], TNF1, TNF + contigs[i] * 136);
        if (dist > local_max) {
            local_max = dist;
        }
    }
    shared_max[threadIdx.x] = local_max;
    __syncthreads();
    // reduction max
    for (size_t i = blockDim.x / 2; i > 0; i >>= 1) {
        if (threadIdx.x < i) {
            if (shared_max[threadIdx.x] < shared_max[threadIdx.x + i]) {
                shared_max[threadIdx.x] = shared_max[threadIdx.x + i];
            }
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        max_dist[contig_idx] = 1.0 - (1.0 / (1 + exp(shared_max[0])));
    }
}

/*
__global__ void get_tnf_prob_sample(double* __restrict__ tnf_dist, float* TNF, double* size_log, size_t* contigs, size_t nobs,
                                    size_t _des, const size_t contig_per_thread, const size_t limit) {
    size_t r1;
    size_t r2;
    float TNF1[136];
    size_t tnf_dist_index = (threadIdx.x + blockIdx.x * blockDim.x) * contig_per_thread;
    size_t prob_index = _des + tnf_dist_index;
    r1 = prob_index / nobs;
    r2 = prob_index % nobs;
    size_t _limit2 = min(tnf_dist_index + contig_per_thread, limit - _des);
    if (tnf_dist_index >= _limit2) return;
    while (tnf_dist_index != _limit2) {
        for (int i = 0; i < 136; i++) {
            TNF1[i] = TNF[contigs[r1] * 136 + i];
        }
        while (r2 < nobs) {
            if (tnf_dist_index == _limit2) break;
            tnf_dist[tnf_dist_index] =
                1. - cal_tnf_dist_d(size_log[contigs[r1]], size_log[contigs[r2]], TNF1, TNF + contigs[r2] * 136);
            tnf_dist_index++;
            r2++;
        }
        r2 = 0;
        r1++;
    }
}
*/

__device__ short get_tn(char* __restrict__ contig) {
    unsigned char N;
    short tn = 0;
    // if (contig[0] == 'X') return 256;
    for (short i = 0; i < 4; i++) {
        N = BN[contig[i]];
        if (N & 4) return 256;
        tn = (tn << 2) | N;
    }
    return tn;
}

__device__ void next_contig(char* __restrict__ contig, char c) {
    if (c == '\n') {
        contig[0] = 'X';
        return;
    }
    for (int i = 0; i < 3; i++) {
        contig[i] = contig[i + 1];
    }
    contig[3] = c;
}

__global__ void get_TNF(float* __restrict__ TNF_d, const char* __restrict__ seqs_d, const size_t* __restrict__ seqs_d_index,
                        const size_t nobs, const size_t contigs_per_thread, const size_t seqs_d_index_size) {
    const size_t thead_id = threadIdx.x + blockIdx.x * blockDim.x;
    size_t limit = min(thead_id * contigs_per_thread + contigs_per_thread, nobs);
    for (size_t contig_index = thead_id * contigs_per_thread; contig_index < limit; contig_index++) {
        char contig_temp[4] = {0};
        float TNF_temp[136] = {0};
        const size_t tnf_index = contig_index * 136;
        size_t contig_size = seqs_d_index[contig_index + seqs_d_index_size] - seqs_d_index[contig_index];
        const char* contig = seqs_d + seqs_d_index[contig_index];
        if (contig_size < 4) continue;
        for (size_t j = 0; j < 3; j++) next_contig(contig_temp, contig[j]);
        for (size_t j = 3; j < contig_size; ++j) {
            char c = contig[j];
            next_contig(contig_temp, c);
            short tn = get_tn(contig_temp);
            if (tn & 256) continue;
            TNF_temp[TNmap_d[tn]]++;
        }
        double rsum = 0;
        for (int c = 0; c < 136; ++c) {
            rsum += TNF_temp[c] * TNF_temp[c];
        }
        rsum = sqrt(rsum);
        for (int c = 0; c < 136; ++c) {
            TNF_d[tnf_index + c] = TNF_temp[c] / rsum;
        }
    }
}

void getError(std::string s = "") {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        hipDeviceReset();
        if (s != "") std::cerr << s << "|";
        std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
            exit(1);
        }
        exit(1);
    }
}

void launch_tnf_kernel(size_t cobs, size_t _first, size_t global_des) {
    size_t n_STREAMS = ((cobs + 9999) / 10000);
    hipStream_t streams[n_STREAMS];
    hipMalloc((void**)&seqs_d, seqs_h_index_e[cobs - 1] * sizeof(char));
    hipMalloc((void**)&seqs_d_index, 2 * cobs * sizeof(size_t));
    size_t contig_per_kernel = cobs / n_STREAMS;
    for (int i = 0; i < n_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
        size_t contig_to_process = contig_per_kernel;
        size_t _des = contig_per_kernel * i;
        size_t TNF_des = _des * 136;
        if (i == n_STREAMS - 1) contig_to_process += (cobs % n_STREAMS);
        size_t bloqs = (contig_to_process + numThreads2 - 1) / numThreads2;
        size_t contigs_per_thread = 1;
        hipMemcpyAsync(seqs_d + seqs_h_index_i[_des], seqs[_first].data() + seqs_h_index_i[_des],
                        seqs_h_index_e[_des + contig_to_process - 1] - seqs_h_index_i[_des], hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(seqs_d_index + _des, seqs_h_index_i.data() + _des, contig_to_process * sizeof(size_t), hipMemcpyHostToDevice,
                        streams[i]);
        hipMemcpyAsync(seqs_d_index + cobs + _des, seqs_h_index_e.data() + _des, contig_to_process * sizeof(size_t),
                        hipMemcpyHostToDevice, streams[i]);
        get_TNF<<<bloqs, numThreads2, 0, streams[i]>>>(TNF_d + 136 * global_des + TNF_des, seqs_d, seqs_d_index + _des,
                                                       contig_to_process, contigs_per_thread, cobs);
        /*
        hipMemcpyAsync(TNF_data + 136 * global_des + TNF_des, TNF_d + 136 * global_des + TNF_des,
                        contig_to_process * 136 * sizeof(float), hipMemcpyDeviceToHost, streams[i]);
        */
    }
    for (int i = 0; i < n_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }
    getError("kernel");
    hipFree(seqs_d);
    hipFree(seqs_d_index);
}

void launch_tnf_max_prob_sample_kernel(std::vector<size_t> idx, double* max_dist_d, double* max_dist_h, size_t _nobs) {
    size_t* contigs_d;
    hipMalloc((void**)&contigs_d, idx.size() * sizeof(size_t));
    hipMemcpy(contigs_d, idx.data(), idx.size() * sizeof(size_t), hipMemcpyHostToDevice);
    get_tnf_max_prob_sample3<<<_nobs, numThreads2, numThreads2 * sizeof(double)>>>(max_dist_d, TNF_d, contig_log, contigs_d, nobs, 0,
                                                                                   _nobs);
    hipDeviceSynchronize();
    hipMemcpy(max_dist_h, max_dist_d, _nobs * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(contigs_d);
    getError("kernel");
}

void reader(int fpint, int id, size_t chunk, size_t _size, char* _mem) {
    size_t readSz = 0;
    while (readSz < _size) {
        size_t _bytesres = _size - readSz;
        readSz += pread(fpint, _mem + (id * chunk) + readSz, _bytesres, (id * chunk) + readSz);
        if (readSz == 0) {
            std::cerr << "Error reading file" << std::endl;
            exit(1);
        }
    }
}

static void print_message(const char* format, ...) {
    va_list argptr;
    va_start(argptr, format);
    vfprintf(stdout, format, argptr);
    cout.flush();
    va_end(argptr);
}

static void verbose_message(const char* format, ...) {
    if (verbose) {
        t2 = std::chrono::steady_clock::now();
        std::chrono::steady_clock::duration duration = t2 - t1;
        int elapsed = (int)std::chrono::duration_cast<std::chrono::seconds>(duration).count();  // seconds
        printf("[%02d:%02d:%02d] ", elapsed / 3600, (elapsed % 3600) / 60, elapsed % 60);
        va_list argptr;
        va_start(argptr, format);
        vfprintf(stdout, format, argptr);
        cout.flush();
        va_end(argptr);
    }
}

class Graph {
   public:
    size_t n;
    std::vector<size_t> from;
    std::vector<size_t> to;
    std::vector<std::vector<size_t>> incs;  // incidence list which has edge id instead of node id (compared to adjacent list)
    std::vector<double> sTNF;
    std::vector<double> sSCR;  // composite score (weight) of sTNF and sABD
    ContigSet connected_nodes;
    bool hasEdges;
    Graph(size_t num_nodes, bool hasEdges = false) : n(num_nodes), hasEdges(hasEdges) {
        if (hasEdges) {
            incs.resize(num_nodes);
        }
    }
    ~Graph() {}
    size_t getNodeCount() { return n; }
    size_t getEdgeCount() { return from.size(); }
    size_t getOtherNode(size_t e, size_t v) {
        assert(e < from.size() && e < to.size());
        return from[e] == v ? to[e] : from[e];
    }
};

static void trim_fasta_label(std::string& label) {
    size_t pos = label.find_first_of(" \t");
    if (pos != std::string::npos) label = label.substr(0, pos);
}

std::ostream& printFasta(std::ostream& os, std::string_view label, std::string_view seq) {
    int64_t len = seq.size();
    if (len == 0) {
        cerr << "Warning attempt to print an empty fasta!" << endl;
        return os;
    }
    os << fasta_delim << label << line_delim;
    const char* _seq = seq.begin();
    const int maxWidth = 60;
    for (size_t s = 0; s < len; s += maxWidth) {
        int bytes = s + maxWidth < len ? maxWidth : len - s;
        os.write(_seq + s, bytes);
        os << line_delim;
    }
    return os;
}

// Fisher-Yates shuffle
// http://stackoverflow.com/questions/9345087/choose-m-elements-randomly-from-a-vector-containing-n-elements
template <class bidiiter>
bidiiter random_unique(bidiiter begin, bidiiter end, size_t num_random) {
    size_t left = std::distance(begin, end);
    while (num_random--) {
        bidiiter r = begin;
        std::advance(r, rand() % left);
        std::swap(*begin, *r);
        ++begin;
        --left;
    }
    return begin;
}

#ifdef __APPLE__
vm_statistics_data_t vmStats;
mach_msg_type_number_t infoCount = HOST_VM_INFO_COUNT;
#else
struct sysinfo memInfo;
#endif
double totalPhysMem = 0.;

int parseLine(char* line) {
    int i = strlen(line);
    while (*line < '0' || *line > '9') line++;
    line[i - 3] = '\0';
    i = atoi(line);
    return i;
}

double getTotalPhysMem() {
    if (totalPhysMem < 1) {
#ifdef __APPLE__
        kern_return_t kernReturn = host_statistics(mach_host_self(), HOST_VM_INFO, (host_info_t)&vmStats, &infoCount);
        if (kernReturn != KERN_SUCCESS) return 0;
        return (vm_page_size * (vmStats.wire_count + vmStats.active_count + vmStats.inactive_count + vmStats.free_count)) / 1024;
#else
        sysinfo(&memInfo);
        long long _totalPhysMem = memInfo.totalram;
        _totalPhysMem *= memInfo.mem_unit;
        totalPhysMem = (double)_totalPhysMem / 1024;  // kb
#endif
    }
    return totalPhysMem;
}

// http://blog.csdn.net/hengshan/article/details/9201929
int getFreeMem() {
#ifdef __APPLE__
    kern_return_t kernReturn = host_statistics(mach_host_self(), HOST_VM_INFO, (host_info_t)&vmStats, &infoCount);
    if (kernReturn != KERN_SUCCESS) return 0;
    return (vm_page_size * vmStats.free_count) / 1024;
#else
    FILE* file = fopen("/proc/meminfo", "r");
    size_t result = 0;
    char line[128];

    while (fgets(line, 128, file) != NULL) {
        if (strncmp(line, "MemFree:", 6) == 0 || strncmp(line, "Buffers:", 6) == 0 || strncmp(line, "Cached:", 6) == 0 ||
            strncmp(line, "SwapFree:", 6) == 0) {
            result += parseLine(line);
        }
    }
    fclose(file);
    return result;  // Kb
#endif
}

double getUsedPhysMem() { return (getTotalPhysMem() - getFreeMem()) / 1024. / 1024.; }

int label_propagation(Graph& g, std::vector<size_t>& membership, std::vector<size_t>& node_order) {
    size_t no_of_nodes = g.getNodeCount();
    size_t no_of_edges = g.getEdgeCount();

    if (no_of_nodes == 0 || no_of_edges == 0) {
        cerr << "There were " << no_of_nodes << " nodes and " << no_of_edges << " edges -- skipping label_propagation" << endl;
        return 0;
    }

    if (g.sSCR.size() != no_of_edges) {
        cerr << "sSCR != no_of_edges" << endl;
        exit(1);
    }

    if (membership.size() != no_of_nodes) {
        membership.resize(no_of_nodes);
        std::iota(membership.begin(), membership.end(), 0);
    }
    /* Do some initial checks */
    if (*std::min_element(g.sSCR.begin(), g.sSCR.end()) < 0) {
        cerr << "sSCR must be non-negative" << endl;
        exit(1);
    }
    std::unordered_map<size_t, std::unordered_set<size_t>> visited;
    std::unordered_set<size_t> blacklist;
    size_t nLeftMin = INT_MAX;
    size_t attempt = 0;
    bool running = true;
    while (running) {
        running = false;
        size_t nLeft = 0;
        /* In the prescribed order, loop over the vertices and reassign labels */
        for (size_t i = 0; i < node_order.size();
             i++) {  // we reconsider all nodes regardless of its previous status, but is it better?
            size_t v1 = node_order[i];
            std::unordered_map<size_t, double> neighbor_scores;  // sum of neighbors scores to cluster k
            std::unordered_map<size_t, size_t> neighbor_counts;  // keep number of neighbors
            std::vector<size_t>& ineis = g.incs[v1];
            for (size_t j = 0; j < ineis.size(); j++) {  // # of neighbors (edges connected to v1)
                size_t edgeID = ineis[j];
                int_fast32_t k = membership[g.getOtherNode(edgeID, v1)];  // community membership of a neighbor (connected by j)
                if (neighbor_scores.find(k) == neighbor_scores.end()) {
                    neighbor_scores[k] = 0.;
                    neighbor_counts[k] = 0;
                }
                neighbor_scores[k] += LOG(1. - g.sSCR[edgeID]);  // as p-value
                neighbor_counts[k]++;
            }

            if (neighbor_scores.size() > 0) {
                for (auto& kv : neighbor_scores) {
                    // Fisher's method to compare significance of different number of probs.
                    boost::math::chi_squared chi_sqr_dist(2 * neighbor_counts[kv.first]);
                    kv.second = boost::math::cdf(chi_sqr_dist, -2.0 * kv.second);
                }
                auto best_neighbor = std::max_element(
                    neighbor_scores.begin(), neighbor_scores.end(),
                    [](const std::pair<size_t, double>& p1, const std::pair<size_t, double>& p2) { return p1.second < p2.second; });
                // however, if there was a clique (loop) out of >2 nodes
                int kPrev = membership[v1];
                if (kPrev != (int)best_neighbor->first && blacklist.find(v1) == blacklist.end()) {
                    membership[v1] = best_neighbor->first;
                    int kNext = membership[v1];
                    if (visited.find(v1) == visited.end() || visited[v1].find(kNext) == visited[v1].end()) {
                        // not have been assigned to the cls before
                        nLeft++;  // # of confirmation (that this choice is optimal) left
                        running = true;
                    } else {
                        blacklist.insert(v1);  // blacklist represents nodes that change cls in a circular form
                    }
                    visited[v1].insert(kNext);
                }
            }
        }

        if (nLeft < nLeftMin) {
            nLeftMin = nLeft;
            attempt = 0;
        } else {
            attempt++;
            if (attempt >= 10) {
                break;
            }
        }
        // cout << "nLeft: " << nLeft << " & attempt: " << attempt << endl;
    }

    return 0;
}

float get_element(boost::numeric::ublas::matrix<float> const& m, int i, int j) { return m(i, j); }

struct CompareEdge {
    constexpr bool operator()(std::pair<size_t, Similarity> const& a, std::pair<size_t, Similarity> const& b) const noexcept {
        return a.second > b.second;
    }
};

// for normal distributions
Distance cal_abd_dist2(Normal& p1, Normal& p2) {
    Distance k1, k2, tmp, d = 0;
    Distance m1 = p1.mean();
    Distance m2 = p2.mean();
    Distance v1 = p1.standard_deviation();
    v1 = v1 * v1;
    Distance v2 = p2.standard_deviation();
    v2 = v2 * v2;

    // normal_distribution
    if (FABS(v2 - v1) < 1e-4) {
        k1 = k2 = (m1 + m2) / 2;
    } else {
        tmp = SQRT(v1 * v2 * ((m1 - m2) * (m1 - m2) - 2 * (v1 - v2) * LOG(SQRT(v2 / v1))));
        k1 = (tmp - m1 * v2 + m2 * v1) / (v1 - v2);
        k2 = (tmp + m1 * v2 - m2 * v1) / (v2 - v1);
    }

    if (k1 > k2) {
        tmp = k1;
        k1 = k2;
        k2 = tmp;
    }
    if (v1 > v2) {
        std::swap(p1, p2);
    }

    if (k1 == k2)
        d = (FABS(boost::math::cdf(p1, k1) - boost::math::cdf(p2, k1)));
    else
        d = (FABS(boost::math::cdf(p1, k2) - boost::math::cdf(p1, k1) + boost::math::cdf(p2, k1) - boost::math::cdf(p2, k2)));

    return d;
}

Distance cal_abd_dist(size_t r1, size_t r2, size_t i, bool& nz) {
    Distance d = 0;
    Distance m1 = ABD(r1, i);
    Distance m2 = ABD(r2, i);
    if (m1 > minCV || m2 > minCV) {
        nz = true;
        m1 = std::max(m1, (Distance)1e-6);
        m2 = std::max(m2, (Distance)1e-6);
        if (m1 != m2) {
            Distance v1 = ABD_VAR(r1, i) < 1 ? 1 : ABD_VAR(r1, i);
            Distance v2 = ABD_VAR(r2, i) < 1 ? 1 : ABD_VAR(r2, i);

            Normal p1(m1, SQRT(v1)), p2(m2, SQRT(v2));
            d = cal_abd_dist2(p1, p2);
        }
    }
    return std::min(std::max(d, 1e-6), 1. - 1e-6);
}

/*
Distance cal_tnf_dist(size_t r1, size_t r2) {
    // EXP(preProb) <= 9 yields prob >= 0.1, so preProb <= LOG(9.0);
    const Distance floor_prob = 0.1;
    const Distance floor_preProb = LOG((1.0 / floor_prob) - 1.0);
    Distance d = 0;
    for (size_t i = 0; i < nTNF; ++i) {
        d += (TNF(r1, i) - TNF(r2, i)) * (TNF(r1, i) - TNF(r2, i));  // euclidean distance
    }
    d = SQRT(d);
    Distance b, c;  // parameters
    Distance ctg1 = logSizes[r1];
    Distance ctg2 = logSizes[r2];
    Distance lw11 = std::min(ctg1, ctg2);
    Distance lw21 = std::max(ctg1, ctg2);
    Distance lw12 = lw11 * lw11;
    Distance lw13 = lw12 * lw11;
    Distance lw14 = lw13 * lw11;
    Distance lw15 = lw14 * lw11;
    Distance lw16 = lw15 * lw11;
    Distance lw17 = lw16 * lw11;
    Distance lw22 = lw21 * lw21;
    Distance lw23 = lw22 * lw21;
    Distance lw24 = lw23 * lw21;
    Distance lw25 = lw24 * lw21;
    Distance lw26 = lw25 * lw21;
    Distance prob;
    b = 46349.1624324381 + -76092.3748553155 * lw11 + -639.918334183 * lw21 + 53873.3933743949 * lw12 + -156.6547554844 * lw22 +
        -21263.6010657275 * lw13 + 64.7719132839 * lw23 + 5003.2646455284 * lw14 + -8.5014386744 * lw24 + -700.5825500292 * lw15 +
        0.3968284526 * lw25 + 54.037542743 * lw16 + -1.7713972342 * lw17 + 474.0850141891 * lw11 * lw21 + -23.966597785 * lw12 * lw22 +
        0.7800219061 * lw13 * lw23 + -0.0138723693 * lw14 * lw24 + 0.0001027543 * lw15 * lw25;
    c = -443565.465710869 + 718862.10804858 * lw11 + 5114.1630934534 * lw21 + -501588.206183097 * lw12 + 784.4442123743 * lw22 +
        194712.394138513 * lw13 + -377.9645994741 * lw23 + -45088.7863182741 * lw14 + 50.5960513287 * lw24 + 6220.3310639927 * lw15 +
        -2.3670776453 * lw25 + -473.269785487 * lw16 + 15.3213264134 * lw17 + -3282.8510348085 * lw11 * lw21 +
        164.0438603974 * lw12 * lw22 + -5.2778800755 * lw13 * lw23 + 0.0929379305 * lw14 * lw24 + -0.0006826817 * lw15 * lw25;

    // logistic model
    //  prob = 1.0 / (1 + EXP(-(b + c * d)));
    //  if (prob >= .1)  //second logistic model
    Distance preProb = -(b + c * d);
    // preProb <= LOG(9.0) yields prob > 0.1, so use second logistic model
    prob = preProb <= floor_preProb ? floor_prob : 1.0 / (1 + EXP(preProb));

    if (prob >= floor_prob) {  // second logistic model
        b = 6770.9351457442 + -5933.7589419767 * lw11 + -2976.2879986855 * lw21 + 3279.7524685865 * lw12 + 1602.7544794819 * lw22 +
            -967.2906583423 * lw13 + -462.0149190219 * lw23 + 159.8317289682 * lw14 + 74.4884405822 * lw24 + -14.0267151808 * lw15 +
            -6.3644917671 * lw25 + 0.5108811613 * lw16 + 0.2252455343 * lw26 + 0.965040193 * lw12 * lw22 +
            -0.0546309127 * lw13 * lw23 + 0.0012917084 * lw14 * lw24 + -1.14383e-05 * lw15 * lw25;
        c = 39406.5712626297 + -77863.1741143294 * lw11 + 9586.8761567725 * lw21 + 55360.1701572325 * lw12 + -5825.2491611377 * lw22 +
            -21887.8400068324 * lw13 + 1751.6803621934 * lw23 + 5158.3764225203 * lw14 + -290.1765894829 * lw24 +
            -724.0348081819 * lw15 + 25.364646181 * lw25 + 56.0522105105 * lw16 + -0.9172073892 * lw26 + -1.8470088417 * lw17 +
            449.4660736502 * lw11 * lw21 + -24.4141920625 * lw12 * lw22 + 0.8465834103 * lw13 * lw23 + -0.0158943762 * lw14 * lw24 +
            0.0001235384 * lw15 * lw25;
        // prob = 1.0 / (1 + EXP(-(b + c * d)));
        //  prob = prob < .1 ? .1 : prob;
        preProb = -(b + c * d);  // EXP(preProb) <= 9 yields prob >= 0.1, so preProb <= LOG(9.0) to calculate, otherwise use the floor
        prob = preProb <= floor_preProb ? 1.0 / (1 + EXP(preProb)) : floor_prob;
    }

    return prob;
}
*/

size_t countLines(const char* f) {
    size_t lines = 0;
    FILE* pFile;
    pFile = fopen(f, "r");
    if (pFile == NULL) {
        cerr << "[Error!] can't open input file " << f << endl;
        return 0;
    }
    while (EOF != fscanf(pFile, "%*[^\n]") && EOF != fscanf(pFile, "%*c")) ++lines;
    fclose(pFile);
    return lines;
}

size_t ncols(std::ifstream& is, int skip = 0) {
    size_t nc = 0;
    std::string firstLine;
    while (skip-- >= 0) std::getline(is, firstLine);
    std::stringstream ss(firstLine);
    std::string col;
    while (std::getline(ss, col, tab_delim)) {
        ++nc;
    }
    return nc;
}

size_t ncols(const char* f, int skip = 0) {
    std::ifstream is(f);
    if (!is.is_open()) {
        cerr << "[Error!] can't open input file " << f << endl;
        return 0;
    }

    return ncols(is, skip);
}

// refer to http://stackoverflow.com/questions/6089231/getting-std-ifstream-to-handle-lf-cr-and-crlf
std::istream& safeGetline(std::istream& is, std::string& t) {
    t.clear();
    // The characters in the stream are read one-by-one using a std::streambuf.
    // That is faster than reading them one-by-one using the std::istream.
    // Code that uses streambuf this way must be guarded by a sentry object.
    // The sentry object performs various tasks,
    // such as thread synchronization and updating the stream state.
    std::istream::sentry se(is, true);
    std::streambuf* sb = is.rdbuf();

    for (;;) {
        int c = sb->sbumpc();
        switch (c) {
            case '\n':
                return is;
            case '\r':
                if (sb->sgetc() == '\n') sb->sbumpc();
                return is;
            case EOF:
                // Also handle the case when the last line has no line ending
                if (t.empty()) is.setstate(std::ios::eofbit);
                return is;
            default:
                t += (char)c;
        }
    }
}

bool is_nz(size_t r1, size_t r2) {
    if (abdFile.empty()) return true;
    Distance _minCV = 1;
    for (size_t i = 0; i < nABD; ++i) {
        Distance m1 = ABD(r1, i);
        Distance m2 = ABD(r2, i);
        if (m1 > _minCV || m2 > _minCV) {  // compare only at least one >2
            return true;
        }
    }
    return false;
}

#pragma omp declare reduction(merge_size_t : std::vector<size_t> : omp_out.insert(omp_out.end(), omp_in.begin(), omp_in.end()))
#pragma omp declare reduction(merge_double : std::vector<double> : omp_out.insert(omp_out.end(), omp_in.begin(), omp_in.end()))

void gen_tnf_graph(Graph& g, Similarity cutoff) {
    ProgressTracker progress = ProgressTracker(nobs);
    std::vector<size_t>& from = g.from;
    std::vector<size_t>& to = g.to;
    std::vector<double>& sTNF = g.sTNF;
    size_t TILE = 10;
    try {
        TILE = std::max(
            (size_t)((CacheSize() * 1024.) / (2 * sizeof(float) * nTNF + maxEdges * (2 * sizeof(size_t) + 1 * sizeof(double)))),
            (size_t)10);
    } catch (...) {
    }
    // printf("TILE: %d\n", TILE);

    const double floor_preProb_cutoff = log((1.0 / (1. - cutoff)) - 1.0);
    // #pragma omp parallel for schedule(dynamic, 1) proc_bind(spread) reduction(merge_size_t: from) reduction(merge_size_t: to)
    // reduction(merge_double: sTNF)

#pragma omp parallel for schedule(dynamic, 1) reduction(merge_size_t : from) reduction(merge_size_t : to) \
    reduction(merge_double : sTNF)
    for (size_t ii = 0; ii < nobs; ii += TILE) {
        double *graph_d, *graph_h;
        hipMalloc((void**)&graph_d, TILE * TILE * sizeof(double));
        hipHostMalloc((void**)&graph_h, TILE * TILE * sizeof(double));
        std::vector<std::priority_queue<std::pair<size_t, double>, std::vector<std::pair<size_t, double>>, CompareEdge>> edges(TILE);
        size_t matrix_y = min(TILE, (nobs - ii));
        for (size_t jj = 0; jj < nobs; jj += TILE) {
            size_t matrix_x = min(TILE, (nobs - jj));
            TIMERSTART(1);
            if (jj == 0) {
                size_t bloqs = ((matrix_x * matrix_y) + numThreads2 - 1) / numThreads2;
                get_tnf_graph<<<bloqs, numThreads2>>>(graph_d, TNF_d, contig_log, matrix_y, matrix_x, ii, jj, floor_preProb_cutoff);

                /*
                get_tnf_graph2<<<matrix_y, numThreads2>>>(graph_d, TNF_d, contig_log, matrix_y, matrix_x, ii, jj,
                                                          floor_preProb_cutoff);
                */
            }
            hipDeviceSynchronize();
            TIMERSTOP(1);
            exit(1);

            hipMemcpy(graph_h, graph_d, TILE * matrix_x * sizeof(double), hipMemcpyDeviceToHost);
            if (jj + TILE <= nobs) {
                size_t matrix_next_x = min(TILE, (nobs - jj - TILE));
                size_t bloqs = ((matrix_next_x * matrix_y) + numThreads2 - 1) / numThreads2;
                get_tnf_graph<<<bloqs, numThreads2>>>(graph_d, TNF_d, contig_log, matrix_y, matrix_next_x, ii, jj + TILE,
                                                      floor_preProb_cutoff);
                /*
                get_tnf_graph2<<<matrix_y, numThreads2>>>(graph_d, TNF_d, contig_log, matrix_y, matrix_next_x, ii, jj + TILE,
                                                          floor_preProb_cutoff);
                */
            }
            for (size_t i = ii; i < ii + TILE && i < nobs; ++i) {
                size_t que_index = i - ii;
                size_t graph_des = que_index * matrix_x;
                for (size_t j = jj; j < jj + TILE && j < nobs; ++j) {
                    if (i == j || !is_nz(i, j)) continue;
                    double sTNF = graph_h[graph_des + (j - jj)];
                    if (sTNF && (edges[que_index].size() < maxEdges ||
                                 (edges[que_index].size() == maxEdges && sTNF > edges[que_index].top().second))) {
                        if (edges[que_index].size() == maxEdges) edges[que_index].pop();
                        edges[que_index].push(std::make_pair(j, sTNF));
                    }
                }
            }
        }
        for (size_t k = 0; k < TILE; ++k) {
            while (!edges[k].empty()) {
                std::pair<size_t, double> edge = edges[k].top();
                if ((ii + k) < edge.first) {
                    sTNF.push_back(edge.second);
                    from.push_back((ii + k));
                    to.push_back(edge.first);
                }
                edges[k].pop();
            }
        }
        if (verbose) {
            progress.track(TILE);
            if (omp_get_thread_num() == 0 && progress.isStepMarker()) {
                verbose_message("Building TNF Graph %s [%.1fGb / %.1fGb]                           \r", progress.getProgress(),
                                getUsedPhysMem(), getTotalPhysMem() / 1024 / 1024);
            }
        }
        hipFree(graph_d);
        hipHostFree(graph_h);
    }

    verbose_message("Finished Building TNF Graph (%d edges) [%.1fGb / %.1fGb]                                          \n",
                    g.getEdgeCount(), getUsedPhysMem(), getTotalPhysMem() / 1024 / 1024);

    g.sTNF.shrink_to_fit();
    g.to.shrink_to_fit();
    g.from.shrink_to_fit();
}

size_t gen_tnf_graph_sample(double coverage = 1., bool full = false) {
    size_t _nobs = full ? nobs : std::min(nobs, (size_t)2500);

    // std::vector<unsigned char> connected_nodes;
    // connected_nodes.resize(_nobs);

    std::vector<size_t> idx(nobs);
    std::iota(idx.begin(), idx.end(), 0);
    random_unique(idx.begin(), idx.end(), _nobs);

    double *max_nobs_d, *max_nobs_h;
    hipHostMalloc((void**)&max_nobs_h, _nobs * sizeof(double));
    hipMalloc((void**)&max_nobs_d, _nobs * sizeof(double));
    launch_tnf_max_prob_sample_kernel(idx, max_nobs_d, max_nobs_h, _nobs);
    getError("launch_tnf_max_prob_sample_kernel");
    hipFree(max_nobs_d);

    std::sort(max_nobs_h, max_nobs_h + _nobs, std::greater<double>());

    size_t p = 999, pp = 1000;
    double cov = 0, pcov = 0;
    int round = 0, counton = 0;

    for (; p > 700;) {
        round++;

        double cutoff = (double)p / 1000.;

        while (max_nobs_h[counton] >= cutoff && counton < _nobs) counton++;

        cov = (double)counton / _nobs;

        if (cov >= coverage) {
            // previous cov is closer to coverage then choose prev p instead current p
            if (cov - coverage > coverage - pcov) {
                p = pp;
                cov = pcov;
            }

            break;
        } else
            verbose_message("Preparing TNF Graph Building [pTNF = %2.1f; %d / %d (P = %2.2f%%) round %d]               \r",
                            (double)p / 10., counton, _nobs, cov * 100, round);
        pp = p;
        pcov = cov;

        if (p > 990)              // 99.9, 99.6, 99.3, 99.0
            p -= rand() % 3 + 1;  // choose from 1,2,3
        else if (p > 900)         // 98.5, 98, 97.5, ... 90.0
            p -= rand() % 3 + 3;  // choose from 3,4,5
        else                      // 89, 88, 87, ..., 70
            p -= rand() % 3 + 9;  // choose from 9,10,11
    }

    hipHostFree(max_nobs_h);
    return p;
}

void rescue_singletons(ClassMap& cls) {
    // handle singleton bins that are of cluster size themselves
    verbose_message("There are %d bins already\n", cls.size());
    std::unordered_set<size_t> large_unbinned;
    for (auto i = 0; i < nobs; i++) {
        if (sizes[i] >= minClsSize) large_unbinned.insert(i);
        /*
        if (seqs[i].size() >= minClsSize) {
            large_unbinned.insert(i);
        }
        */
    }
    for (auto it = cls.begin(); it != cls.end(); ++it) {
        size_t kk = it->first;
        size_t s = 0, s1 = 0;

        for (auto it2 = cls[kk].begin(); it2 != cls[kk].end(); ++it2) {
            if (*it2 < (int)nobs) {
                if (sizes[*it2] >= minClsSize) large_unbinned.erase(*it2);  // it was binned!
                // if (seqs[*it2].size() >= minClsSize) large_unbinned.erase(*it2);  // it was binned!
            }
        }
    }
    if (verbose && large_unbinned.size() > 0)
        verbose_message("Rescued %d large contig(s) into singleton bin(s)\n", large_unbinned.size());
    for (auto id : large_unbinned) {
        assert(cls.find(id) == cls.end());
        cls[id].push_back(id);
    }
}

void output_bins(ClassMap& cls) {
#pragma omp parallel
    {
#pragma omp single
        {
            Distance binnedSize = 0, binnedSize1 = 0;
            std::vector<size_t> clsMap(nobs + nobs1, 0);

            size_t bin_id = 1;  // start with bin #1
            for (auto it = cls.begin(); it != cls.end(); ++it) {
                size_t kk = it->first;
                assert(kk >= 0);
                size_t s = 0, s1 = 0;
                {
                    const auto& cluster = it->second;  // in new block for compatiblity with old OpenMP standard that does not support
                                                       // references in private vars

                    for (auto it2 = cluster.begin(); it2 != cluster.end(); ++it2) {
                        if (*it2 < (int)nobs) {
                            // s += seqs[*it2].size();
                            s += sizes[*it2];
                        } else {
                            // s1 += small_seqs[*it2 - nobs].size();
                            s1 += small_sizes[*it2 - nobs];
                        }
                    }

                    if (s + s1 < minClsSize) {
                        continue;
                    }

                    for (size_t i = 0; i < cluster.size(); ++i) {
                        assert(cluster[i] < (int)clsMap.size());
                        clsMap[cluster[i]] = kk + 1;
                    }
                }

                binnedSize += s;
                binnedSize1 += s1;

#pragma omp task
                if (!noBinOut) {
                    auto& cluster = it->second;  // in new block for compatiblity with old OpenMP standard that does not support
                                                 // references in private vars
                    std::string outFile_cls = outFile + ".";
                    outFile_cls.append(boost::lexical_cast<std::string>(bin_id));
                    if (!onlyLabel) outFile_cls.append(".fa");
                    std::sort(cluster.begin(), cluster.end());  // deterministic ordering of contigs within bins

                    size_t bases = 0;
                    std::ofstream os(outFile_cls.c_str());
                    if (!os) {
                        cerr << "[Error!] Could not write to " << outFile_cls << endl;
                        exit(1);
                    }
                    char os_buffer[buf_size];
                    os.rdbuf()->pubsetbuf(os_buffer, buf_size);
                    for (auto it2 = cluster.begin(); it2 != cluster.end(); ++it2) {
                        std::string_view& label = (*it2 < (int)nobs) ? contig_names[*it2] : small_contig_names[*it2 - nobs];
                        if (onlyLabel) {
                            os << label << line_delim;
                        } else {
                            std::string_view& seq = (*it2 < (int)nobs) ? seqs[*it2] : small_seqs[*it2 - nobs];
                            printFasta(os, label, seq);
                            bases += seq.size();
                        }
                    }
                    os.close();
                    if (!os) {
                        cerr << "[Error!] Failed to write to " << outFile_cls << endl;
                        exit(1);
                    }

                    if (debug)
                        cout << "Bin " << bin_id << " (" << bases << " bases in " << cluster.size()
                             << " contigs) was saved to: " << outFile_cls << endl;
                }

                bin_id++;
            }

            if (saveCls) {
#pragma omp task
                {
                    if (verbose) verbose_message("Saving cluster membership matrix to %s\n", outFile.c_str());

                    std::ofstream os(outFile.c_str());
                    if (!os) {
                        cerr << "[Error!] Could not write cluster membership to " << outFile << endl;
                        exit(1);
                    }
                    char os_buffer[buf_size];
                    os.rdbuf()->pubsetbuf(os_buffer, buf_size);

                    for (size_t i = 0; i < nobs; ++i) {
                        os << contig_names[i] << tab_delim << clsMap[i] << line_delim;
                    }
                    for (size_t i = nobs; i < nobs + nobs1; ++i) {
                        os << small_contig_names[i - nobs] << tab_delim << clsMap[i] << line_delim;
                    }

                    os.flush();
                    os.close();
                    if (!os) {
                        cerr << "[Error!] Failed to write cluster membership to " << outFile << endl;
                        exit(1);
                    }
                }
            }

            if (outUnbinned) {
#pragma omp task
                {
                    std::string outFile_cls = outFile + ".";
                    outFile_cls.append("unbinned");
                    if (!onlyLabel) outFile_cls.append(".fa");

                    if (verbose) verbose_message("Saving unbinned contigs to %s\n", outFile_cls.c_str());

                    std::ofstream os(outFile_cls.c_str());
                    if (!os) {
                        cerr << "[Error!] Could not to write unbinned contigs to " << outFile_cls << endl;
                        exit(1);
                    }
                    char os_buffer[buf_size];
                    os.rdbuf()->pubsetbuf(os_buffer, buf_size);

                    for (size_t i = 0; i < clsMap.size(); ++i) {
                        if (clsMap[i] == 0) {
                            std::string_view& label = ((i < nobs) ? contig_names[i] : small_contig_names[i - nobs]);
                            if (onlyLabel) {
                                os << label << line_delim;
                            } else {
                                std::string_view& seq = (i < nobs) ? seqs[i] : small_seqs[i - nobs];
                                printFasta(os, label, seq);
                            }
                        }
                    }
                    os.flush();
                    os.close();
                    if (!os) {
                        cerr << "[Error!] Failed to write unbinned contigs to " << outFile_cls << endl;
                        exit(1);
                    }
                }
            }

#pragma omp taskwait
            if (verbose) {
                verbose_message("%2.2f%% (%lld bases) of large (>=%d) and %2.2f%% (%lld bases) of small (<%d) contigs were binned.\n",
                                (double)binnedSize / totalSize * 100, (unsigned long long)binnedSize, minContig,
                                binnedSize1 == 0 ? 0 : (double)binnedSize1 / totalSize1 * 100, (unsigned long long)binnedSize1,
                                minContig);
            }
            cout.precision(20);
            cout << bin_id - 1 << " bins (" << binnedSize + binnedSize1 << " bases in total) formed." << std::endl;

        }  // omp single
    }      // omp parallel
}

Distance cal_abd_corr(size_t r1, size_t r2, bool is_small = false) {
    size_t i, ii;
    double sum_xsq = 0.0;
    double sum_ysq = 0.0;
    double sum_cross = 0.0;
    double ratio;
    double delta_x, delta_y;
    double mean_x = 0.0, mean_y = 0.0;
    double r = 0.0;
    size_t s = 0;  // skipped
    for (i = 0; i < nABD; ++i) {
        Distance m1 = ABD(r1, i);
        Distance m2 = is_small ? small_ABD(r2, i) : ABD(r2, i);
        ii = i - s;
        if (ii == 0) {
            mean_x = m1;
            mean_y = m2;
            continue;
        }
        ratio = ii / (ii + 1.0);
        delta_x = m1 - mean_x;
        delta_y = m2 - mean_y;
        sum_xsq += delta_x * delta_x * ratio;
        sum_ysq += delta_y * delta_y * ratio;
        sum_cross += delta_x * delta_y * ratio;
        mean_x += delta_x / (ii + 1.0);
        mean_y += delta_y / (ii + 1.0);
    }
    r = sum_cross / (sqrt(sum_xsq) * sqrt(sum_ysq));
    return r;
}

int main(int ac, char* av[]) {
    po::options_description desc("Allowed options", 110, 110 / 2);
    desc.add_options()("help,h", "produce help message")("inFile,i", po::value<std::string>(&inFile),
                                                         "Contigs in (gzipped) fasta file format [Mandatory]")(
        "outFile,o", po::value<std::string>(&outFile),
        "Base file name and path for each bin. The default output is fasta format. Use -l option to output only contig names "
        "[Mandatory].")("abdFile,a", po::value<std::string>(&abdFile),
                        "A file having mean and variance of base coverage depth (tab delimited; the first column should be contig "
                        "names, and the first row will be considered as the header and be skipped) [Optional].")(
        "minContig,m", po::value<size_t>(&minContig)->default_value(2500), "Minimum size of a contig for binning (should be >=1500).")(
        "maxP", po::value<Similarity>(&maxP)->default_value(95),
        "Percentage of 'good' contigs considered for binning decided by connection among contigs. The greater, the more sensitive.")(
        "minS", po::value<Similarity>(&minS)->default_value(60),
        "Minimum score of a edge for binning (should be between 1 and 99). The greater, the more specific.")(
        "maxEdges", po::value<size_t>(&maxEdges)->default_value(200),
        "Maximum number of edges per node. The greater, the more sensitive.")(
        "pTNF", po::value<Similarity>(&pTNF)->default_value(0),
        "TNF probability cutoff for building TNF graph. Use it to skip the preparation step. (0: auto).")(
        "noAdd", po::value<bool>(&noAdd)->zero_tokens(), "Turning off additional binning for lost or small contigs.")(
        "cvExt", po::value<bool>(&cvExt)->zero_tokens(),
        "When a coverage file without variance (from third party tools) is used instead of abdFile from "
        "jgi_summarize_bam_contig_depths.")("minCV,x", po::value<Distance>(&minCV)->default_value(1),
                                            "Minimum mean coverage of a contig in each library for binning.")(
        "minCVSum", po::value<Distance>(&minCVSum)->default_value(1),
        "Minimum total effective mean coverage of a contig (sum of depth over minCV) for binning.")(
        "minClsSize,s", po::value<size_t>(&minClsSize)->default_value(200000), "Minimum size of a bin as the output.")(
        "numThreads,t", po::value<size_t>(&numThreads)->default_value(0), "Number of threads to use (0: use all cores).")(
        "onlyLabel,l", po::value<bool>(&onlyLabel)->zero_tokens(),
        "Output only sequence labels as a list in a column without sequences.")("saveCls", po::value<bool>(&saveCls)->zero_tokens(),
                                                                                "Save cluster memberships as a matrix format")(
        "unbinned", po::value<bool>(&outUnbinned)->zero_tokens(), "Generate [outFile].unbinned.fa file for unbinned contigs")(
        "noBinOut", po::value<bool>(&noBinOut)->zero_tokens(),
        "No bin output. Usually combined with --saveCls to check only contig memberships")(
        "seed", po::value<unsigned long long>(&seed)->default_value(0), "For exact reproducibility. (0: use random seed)")(
        "debug,d", po::value<bool>(&debug)->zero_tokens(), "Debug output")(
        "verbose,v", po::value<bool>(&verbose)->zero_tokens(), "Verbose output")("ct", po::value<int>(&numThreads2)->default_value(16),
                                                                                 "Number of cuda threads");

    po::variables_map vm;
    po::store(po::command_line_parser(ac, av).options(desc).positional({}).run(), vm);
    po::notify(vm);

    if (vm.count("help") || inFile.length() == 0 || outFile.length() == 0) {
        cerr << "\nMetaBAT: Metagenome Binning based on Abundance and Tetranucleotide frequency (version 2:" << version << "; " << DATE
             << ")" << endl;
        cerr << "by Don Kang (ddkang@lbl.gov), Feng Li, Jeff Froula, Rob Egan, and Zhong Wang (zhongwang@lbl.gov) \n" << endl;
        cerr << desc << endl << endl;

        if (!vm.count("help")) {
            if (inFile.empty()) {
                cerr << "[Error!] There was no --inFile specified" << endl;
            }
            if (outFile.empty()) {
                cerr << "[Error!] There was no --outFile specified" << endl;
            }
        }

        return vm.count("help") ? 0 : 1;
    }

    if (verbose) t1 = std::chrono::steady_clock::now();

    if (seed == 0) seed = time(0);
    srand(seed);

    if (maxP <= 0 || maxP >= 100) {
        cerr << "[Error!] maxP should be greater than 0 and less than 100" << endl;
        return 1;
    }

    if (minS <= 1 || minS >= 100) {
        cerr << "[Error!] minS should be greater than 1 and less than 100" << endl;
        return 1;
    }

    if (pTNF < 0 || pTNF >= 100) {
        cerr << "[Error!] pTNF should be >= 0 and < 100" << endl;
        return 1;
    }

    if (minContig < 1500) {
        cerr << "[Error!] Contig length < 1500 is not allowed to be used for binning." << endl;
        return 1;
    }

    if (minCV < 0) {
        cerr << "[Error!] minCV should be non-negative" << endl;
        return 1;
    }
    minCVSum = std::max(minCV, minCVSum);

    boost::filesystem::path dir(outFile);
    boost::system::error_code ec;
    if (dir.parent_path().string().length() > 0) {
        if (boost::filesystem::is_regular_file(dir.parent_path())) {
            cerr << "Cannot create directory: " << dir.parent_path().string() << ", which exists as a regular file." << endl;
            return 1;
        }
        if (!boost::filesystem::is_directory(dir.parent_path()) && !boost::filesystem::create_directory(dir.parent_path(), ec)) {
            cerr << "Cannot create directory: " << dir.parent_path().string() << ": " << ec << endl;
            return 1;
        }
    }

    print_message(
        "MetaBAT 2 (%s) using minContig %d, minCV %2.1f, minCVSum %2.1f, maxP %2.0f%%, minS %2.0f, maxEdges %d and minClsSize %d. "
        "with random seed=%lld\n",
        version.c_str(), minContig, minCV, minCVSum, maxP, minS, maxEdges, minClsSize, seed);

    maxP /= 100., minS /= 100.;

    if (numThreads == 0)
        numThreads = omp_get_max_threads();
    else
        numThreads = std::min(numThreads, (size_t)omp_get_max_threads());

    if (numThreads2 % 32 != 0) {
        cerr << "numThreads2 must be a multiple of 32" << endl;
        return 1;
    }

    omp_set_num_threads(numThreads);
    verbose_message("Executing with %d CPU threads and %d CUDA threads\n", numThreads, numThreads2);

    nobs = 0, nobs1 = 0;

    std::unordered_map<std::string_view, size_t> contigs;
    std::unordered_map<std::string_view, size_t> small_contigs;

    const int nNonFeat = cvExt ? 1 : 3;  // number of non features
    bool hasABD = abdFile.length() > 0;

    // validate fasta and depths file (abd) have same set of sequence identifiers (in same ordering)
    // todo read fasta first, then read depths, then validate
    {
        // need to handle the case where more data in assembly.fa than depth.txt (but contigs should be in order)
        std::unordered_set<std::string> inDepth;

        // todo refactor into validate depths file method
        if (hasABD) {
            verbose_message("Parsing abundance file\n");
            if (countLines(abdFile.c_str()) < 2) {  // the first row is header
                cerr << "[Error!] There are no lines in the abundance depth file or fasta file!" << endl;
                exit(1);
            }
            nABD = ncols(abdFile.c_str(), 1) - nNonFeat;  // num of features (excluding the first three columns which is the
                                                          // contigName, contigLen, and totalAvgDepth);
            if (!cvExt) {
                if (nABD % 2 != 0) {
                    cerr << "[Error!] Number of columns (excluding the first column) in abundance data file is not even." << endl;
                    exit(1);
                }
                nABD /= 2;
            }

            std::ifstream is(abdFile.c_str());
            if (!is.is_open()) {
                cerr << "[Error!] can't open the contig coverage depth file " << abdFile << endl;
                return 1;
            }

            int r = -1;
            for (std::string row; safeGetline(is, row) && is.good(); ++r) {
                if (r == -1)  // the first row is header
                    continue;
                std::stringstream ss(row);
                int c = -nNonFeat;
                for (std::string col; getline(ss, col, tab_delim); ++c) {
                    if (c == -3 || (cvExt && c == -1)) {  // contig name
                        if (col.empty()) break;
                        trim_fasta_label(col);
                        inDepth.insert(col);
                        break;
                    }
                }
            }
            is.close();
        }

        verbose_message("Parsing assembly file\n");

        FILE* fp = fopen(inFile.c_str(), "r");
        if (fp == NULL) {
            cerr << "[Error!] can't open the sequence fasta file " << inFile << endl;
            return 1;
        } else {
            std::ofstream* os = NULL;
            char os_buffer[buf_size];
            std::string filteredFile_cls;
            if (outUnbinned) {
                filteredFile_cls = outFile + ".";
                filteredFile_cls.append("tooShort");
                if (!onlyLabel) {
                    filteredFile_cls.append(".fa");
                }
                os = new std::ofstream(filteredFile_cls.c_str());
                if (!os->is_open() || os->fail() || !*os) {
                    cerr << "[Error!] can't open the output bin file: " << filteredFile_cls << endl;
                    return 1;
                }
                os->rdbuf()->pubsetbuf(os_buffer, buf_size);
                if (verbose) verbose_message("Outputting contigs that are too short to %s\n", filteredFile_cls.c_str());
            }
            fseek(fp, 0L, SEEK_END);
            fsize = ftell(fp);  // obtener el tamaño del archivo
            fclose(fp);
            size_t chunk = fsize / numThreads;
            hipError_t cudaStatus = hipHostMalloc((void**)&_mem, fsize);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "cuda_Mallo_cHost failed!");
                return 1;
            }
            int fpint = open(inFile.c_str(), O_RDWR | O_CREAT, S_IREAD | S_IWRITE | S_IRGRP | S_IROTH);
            if (fpint == -1) {
                std::cout << "Error opening file: " << inFile << std::endl;
                return 1;
            }
            std::thread readerThreads[numThreads];
            for (int i = 0; i < numThreads; i++) {
                size_t _size;
                if (i != numThreads - 1)
                    _size = chunk;
                else
                    _size = chunk + (fsize % numThreads);
                readerThreads[i] = std::thread(reader, fpint, i, chunk, _size, _mem);
            }
            for (int i = 0; i < numThreads; i++) {  // esperar a que terminen de leer
                readerThreads[i].join();
            }

            close(fpint);

            size_t char_per_thread = (fsize + numThreads - 1) / numThreads;
#pragma omp parallel shared(os)
            {
                int t = omp_get_thread_num();
                size_t nobs_l = 0;
                size_t nobs1_l = 0;
                std::vector<Distance> logSizes_l;
                std::vector<size_t> sizes_l;
                std::vector<size_t> small_sizes_l;
                std::unordered_map<std::string_view, size_t> contigs_l;
                std::unordered_map<std::string_view, size_t> small_contigs_l;
                std::vector<std::string_view> contig_names_l;
                std::vector<std::string_view> small_contig_names_l;
                std::vector<std::string_view> seqs_l;
                std::vector<std::string_view> small_seqs_l;
                size_t contig_name_i;
                size_t contig_i;
                size_t contig_size;
                for (size_t i = t * char_per_thread; i < min(fsize, (t + 1) * char_per_thread); i++) {
                    if (_mem[i] == fasta_delim) {
                        size_t cont_lines = 0;
                        i++;
                        contig_name_i = i;  // guardar el inicio del nombre del contig
                        while (_mem[i] != line_delim) i++;
                        std::string_view name(_mem + contig_name_i, i - contig_name_i);
                        i++;
                        contig_i = i;  // guardar el inicio del contig
                        while (i < fsize && _mem[i] != fasta_delim) {
                            if (_mem[i] == line_delim) cont_lines++;
                            i++;
                        }
                        std::string_view seq(_mem + contig_i, i - contig_i);
                        contig_size = seq.length() - cont_lines;
                        if (contig_size >= (int)minContig) {
                            contigs_l[name] = nobs_l++;
                            contig_names_l.push_back(name);
                            seqs_l.push_back(seq);
                            logSizes_l.push_back(LOG10(std::min(contig_size, (size_t)500000)));
                            sizes_l.push_back(contig_size);
                        } else if (contig_size >= (int)1000) {
                            small_contigs_l[name] = nobs1_l++;
                            small_contig_names_l.push_back(name);
                            small_seqs_l.push_back(seq);
                            small_sizes_l.push_back(contig_size);
                        } else if (os) {
                            if (onlyLabel) {
                                *os << name << line_delim;
                            } else {
                                printFasta(*os, name, seq);
                            }
                        }
                        i--;
                    }
                }
                for (int i = 0; i < numThreads; i++) {
#pragma omp barrier
                    if (i == t) {
                        {
                            for (auto& pair : contigs_l) pair.second += nobs;
                            for (auto& pair : small_contigs_l) pair.second += nobs1;
                            contigs.merge(contigs_l);
                            small_contigs.merge(small_contigs_l);
                            contig_names.insert(contig_names.end(), contig_names_l.begin(), contig_names_l.end());
                            small_contig_names.insert(small_contig_names.end(), small_contig_names_l.begin(),
                                                      small_contig_names_l.end());
                            seqs.insert(seqs.end(), seqs_l.begin(), seqs_l.end());
                            small_seqs.insert(small_seqs.end(), small_seqs_l.begin(), small_seqs_l.end());
                            logSizes.insert(logSizes.end(), logSizes_l.begin(), logSizes_l.end());
                            sizes.insert(sizes.end(), sizes_l.begin(), sizes_l.end());
                            small_sizes.insert(small_sizes.end(), small_sizes_l.begin(), small_sizes_l.end());
                            nobs += nobs_l;
                            nobs1 += nobs1_l;
                        }
                    }
                }
            }
        }
    }

    if (contig_names.size() != contigs.size() || small_contig_names.size() != small_contigs.size()) {
        printf("contig_names: %lu, contigs: %lu, small_contig_names: %lu, small_contigs: %lu\n", contig_names.size(), contigs.size(),
               small_contig_names.size(), small_contigs.size());
        cerr << "[Error!] Need to check whether there are duplicated sequence ids in the assembly file" << endl;
        return 1;
    }
    verbose_message("Number of large contigs >= %d are %d. \n", minContig, nobs);

    if (hasABD) {
        ABD.resize(nobs, nABD);
        ABD_VAR.resize(nobs, nABD);
        small_ABD.resize(nobs1, nABD);

        verbose_message("Reading abundance file\n");
        std::ifstream is(abdFile.c_str());
        if (!is.is_open()) {
            cerr << "[Error!] can't open the contig coverage depth file " << abdFile << endl;
            return 1;
        }

        int r = -1;
        size_t num = 0, num1 = 0, nskip = 0, nskip1 = 0;

        std::ofstream* os = NULL;
        char os_buffer[buf_size];
        std::string filteredFile_cls;
        if (outUnbinned) {
            filteredFile_cls = outFile + ".";
            filteredFile_cls.append("lowDepth");
            if (!onlyLabel) {
                filteredFile_cls.append(".fa");
            }
            os = new std::ofstream(filteredFile_cls.c_str());
            if (!os->is_open() || os->fail() || !*os) {
                cerr << "[Error!] Failed to open to " << filteredFile_cls << endl;
                return 1;
            }
            os->rdbuf()->pubsetbuf(os_buffer, buf_size);
            if (verbose) verbose_message("Outputting contigs that are too low depth to %s\n", filteredFile_cls.c_str());
        }
        // todo refactor reading file
        for (std::string row; safeGetline(is, row) && is.good(); ++r) {
            if (r == -1)  // the first row is header
                continue;

            std::stringstream ss(row);
            int c = -nNonFeat;
            Distance mean = 0, variance, meanSum = 0;
            std::string label;
            bool isLarge = false, isGood = false, isSmall = false;
            for (std::string col; getline(ss, col, tab_delim); ++c) {
                if (c == -3 || (cvExt && c == -1)) {  // contig name
                    if (col.empty()) break;
                    trim_fasta_label(col);
                    label = col;

                    if (contigs.find(label) == contigs.end()) {  // small or additional contigs
                        if (small_contigs.find(label) == small_contigs.end())
                            break;
                        else
                            isSmall = true;
                    } else
                        isLarge = true;

                    if ((isSmall && small_contigs[label] != num1) || (isLarge && contigs[label] != num)) {
                        cerr << "[Error!] the order of contigs in abundance file is not the same as the assembly file: " << label
                             << endl;
                        exit(1);
                    }
                    isGood = true;
                    continue;
                } else if (c == -2) {  // contig length
                    continue;
                } else if (c == -1) {  // mean coverage
                    continue;
                }

                bool checkMean = false, checkVar = false;
                if (cvExt) {  // abd file from 3rd party contains only mean coverage, so assuming variance = mean
                    mean = boost::lexical_cast<Distance>(col.c_str());
                    if (mean >= minCV) {  // FIXME? Issue #68
                        meanSum += mean;
                    }
                    variance = mean;
                    checkMean = true;
                    if (isLarge) {
                        ABD(num - nskip, c) = mean;
                        ABD_VAR(num - nskip, c) = mean;
                    } else {
                        small_ABD(num1 - nskip1, c) = mean;
                    }
                } else {
                    if (c % 2 == 0) {
                        checkMean = true;
                        if (isLarge)
                            mean = ABD(num - nskip, c / 2) = boost::lexical_cast<Distance>(col.c_str());
                        else
                            mean = small_ABD(num1 - nskip1, c / 2) = boost::lexical_cast<Distance>(col.c_str());
                        if (mean >= minCV) {  // FIXME? Issue #68
                            meanSum += mean;
                        }
                    } else {
                        checkVar = true;
                        if (isLarge)
                            variance = ABD_VAR(num - nskip, c / 2) = boost::lexical_cast<Distance>(col.c_str());
                        else
                            variance = boost::lexical_cast<Distance>(col.c_str());
                    }
                }

                if (checkMean) {
                    if (mean > 1e+7) {
                        cerr << "[Warning!] Need to check where the average depth is greater than 1e+7 for the contig " << label
                             << ", column " << c + 1 << endl;
                        isGood = false;
                    }
                    if (mean < 0) {
                        cerr << "[Warning!] Negative coverage depth is not allowed for the contig " << label << ", column " << c + 1
                             << ": " << mean << endl;
                        isGood = false;
                    }
                }

                if (checkVar) {
                    if (variance > 1e+14) {
                        cerr << "[Warning!] Need to check where the depth variance is greater than 1e+14 for the contig " << label
                             << ", column " << c + 1 << endl;
                        isGood = false;
                    }
                    if (variance < 0) {
                        cerr << "[Warning!] Negative variance is not allowed for the contig " << label << ", column " << c + 1 << ": "
                             << variance << endl;
                        isGood = false;
                    }
                }

                if (c == (int)(nABD * (cvExt ? 1 : 2) - 1)) {  // last data of the line, check mean coverage
                    if (meanSum < minCVSum) {
                        if (debug)
                            verbose_message("[Info] Ignored a contig (%s) having effective mean coverage %2.2f < %2.2f \n",
                                            label.c_str(), meanSum, minCVSum);
                        isGood = false;
                    }
                }

                if (!isGood) break;
            }

            if (isGood && (int)nABD != (cvExt ? c : c / 2)) {
                cerr << "[Warning!] Different number of variables for the object for the contig " << label << endl;
                isGood = false;
            }

            if (isLarge) {
                if (!isGood) {
                    ++nskip;
                    if (os) {
                        if (onlyLabel) {
                            *os << contig_names[num] << line_delim;
                        } else {
                            printFasta(*os, contig_names[num], seqs[num]);
                        }
                    }
                    contig_names[num] = "";
                    seqs[num] = "";
                    logSizes[num] = -1;
                    sizes[num] = 0;
                }
                ++num;
            } else if (isSmall) {
                if (!isGood) {
                    ++nskip1;
                    if (os) {
                        if (onlyLabel) {
                            *os << small_contig_names[num1] << line_delim;
                        } else {
                            printFasta(*os, small_contig_names[num1], small_seqs[num1]);
                        }
                    }
                    small_contig_names[num1] = "";
                    small_seqs[num1] = "";
                    small_sizes[num1] = 0;
                }
                ++num1;
            }
        }
        is.close();
        if (os) {
            os->close();
            if (!*os) {
                cerr << "[Error!] Failed to write to " << filteredFile_cls << endl;
                return 1;
            }
            delete os;
        }

        assert(nobs == num && nobs1 == num1);

        nobs = num - nskip;
        nobs1 = num1 - nskip1;

        assert(contigs.size() == nobs + nskip && small_contigs.size() == nobs1 + nskip1);

        contigs.clear();
        small_contigs.clear();

        if (debug) {
            verbose_message("nobs = %d\n", nobs);
            verbose_message("r = %d (num = %d), (nskip = %d) \n", r, num, nskip);
        }
        verbose_message("Finished reading %d contigs and %d coverages from %s\n", r, nABD, abdFile.c_str());

        seqs.erase(std::remove(seqs.begin(), seqs.end(), ""), seqs.end());
        assert(nobs == seqs.size());
        small_seqs.erase(std::remove(small_seqs.begin(), small_seqs.end(), ""), small_seqs.end());
        assert(nobs1 == small_seqs.size());
        contig_names.erase(std::remove(contig_names.begin(), contig_names.end(), ""), contig_names.end());
        assert(nobs == contig_names.size());
        small_contig_names.erase(std::remove(small_contig_names.begin(), small_contig_names.end(), ""), small_contig_names.end());
        assert(nobs1 == small_contig_names.size());
        logSizes.erase(std::remove(logSizes.begin(), logSizes.end(), -1), logSizes.end());
        assert(nobs == logSizes.size());
        sizes.erase(std::remove(sizes.begin(), sizes.end(), 0), sizes.end());
        assert(nobs == sizes.size());
        small_sizes.erase(std::remove(small_sizes.begin(), small_sizes.end(), 0), small_sizes.end());
        assert(nobs1 == small_sizes.size());
        if (debug) {
            verbose_message("seqs.size = %d, contig_names.size = %d\n", seqs.size(), contig_names.size());
        }

        if (ABD.size1() != nobs) {
            ABD.resize(nobs, nABD, true);
            ABD_VAR.resize(nobs, nABD, true);
        }
        if (small_ABD.size1() != nobs1) {
            small_ABD.resize(nobs1, nABD, true);
        }
    }
    verbose_message("Number of target contigs: %d of large (>= %d) and %d of small ones (>=%d & <%d). \n", nobs, minContig, nobs1,
                    1000, minContig);

    // prepare logsizes
    /*
    logSizes.resize(nobs);
#pragma omp parallel for
    for (size_t r = 0; r < nobs; ++r) {
        logSizes[r] = LOG10(std::min(seqs[r].size(), (size_t)500000));
    }
    */

    hipMalloc((void**)&contig_log, nobs * sizeof(double));
    hipMemcpy(contig_log, logSizes.data(), nobs * sizeof(double), hipMemcpyHostToDevice);

    verbose_message("Start TNF calculation. nobs = %zd\n", nobs);

    size_t max_gpu_mem = 4000000000;  // 4gb
    TIMERSTART(TNF_CAL);

    hipMalloc((void**)&TNF_d, nobs * 136 * sizeof(float));
    {
        ProgressTracker progress(nobs);

        seqs_h_index_i.reserve(nobs);
        seqs_h_index_e.reserve(nobs);

        // TNF.resize(nobs, nTNF);
        // TNF.clear();
        size_t cobs = 0;  // current obs
        size_t _first = 0;
        for (size_t i = 0; i < nobs; i++) {
            if (seqs[i].data() - seqs[_first].data() + seqs[i].size() > max_gpu_mem) {
                launch_tnf_kernel(cobs, _first, i - cobs);
                seqs_h_index_i.clear();
                seqs_h_index_e.clear();
                progress.track(cobs);
                verbose_message("Calculating TNF %s\r", progress.getProgress());
                _first = i;
                cobs = 0;
            }
            seqs_h_index_i.emplace_back(seqs[i].data() - seqs[_first].data());
            seqs_h_index_e.emplace_back(seqs[i].data() - seqs[_first].data() + seqs[i].size());
            cobs++;
        }
        if (cobs != 0) {
            launch_tnf_kernel(cobs, _first, nobs - cobs);
            progress.track(cobs);
            verbose_message("Calculating TNF %s\r", progress.getProgress());
            seqs_h_index_i.clear();
            seqs_h_index_e.clear();
        }
    }
    TIMERSTOP(TNF_CAL);

    verbose_message("Finished TNF calculation.                                  \n");

    ClassMap cls;
    do {
        std::vector<size_t> mems;
        {
            Graph g(nobs);
            TIMERSTART(GRAPH_SAMPLE)
            // 1. sampling graph to find minp
            if (pTNF < 1.) {
                if (nobs <= 25000) {
                    pTNF = gen_tnf_graph_sample(maxP, true);
                } else {
                    for (size_t i = 0; i < 10; ++i) {
                        verbose_message("Attempt %d of 10 to gen_tnf_graph_sample\n", i);
                        double _minp = gen_tnf_graph_sample(maxP);
                        verbose_message("\n");
                        if (_minp < 701) _minp = 700.;
                        pTNF += _minp;
                        if (i == 1 && pTNF / 2 < 701) {
                            pTNF = 700.;
                            break;
                        }
                        if (i == 9) pTNF /= 10.;
                    }
                }
            } else {
                pTNF *= 10;
            }
            TIMERSTOP(GRAPH_SAMPLE)
            verbose_message("Finished Preparing TNF Graph Building [pTNF = %2.2f]                                             \n",
                            pTNF / 10.);

            // 2. build tnf graph
            TIMERSTART(TNF_GRAPH);
            gen_tnf_graph(g, pTNF / 1000.);
            TIMERSTOP(TNF_GRAPH);

            size_t nEdges = g.sTNF.size();

            if (nEdges == 0) {
                cout << "No edges were formed by TNF." << endl;
                break;
            }

            // 3. convert sTNF to sSCR
            if (!abdFile.empty()) {
                verbose_message("Applying coverage correlations to TNF graph with %d edges\n", nEdges);
                g.sSCR.resize(nEdges, 0);
                std::vector<double> abd_distr(nEdges);
                std::vector<int> nnz(nEdges, 0);

#pragma omp parallel for
                for (size_t i = 0; i < nEdges; ++i) {
                    for (size_t j = 0; j < nABD; ++j) {
                        bool nz = false;
                        Similarity abd = 1. - cal_abd_dist(g.to[i], g.from[i], j, nz);
                        if (nz) {
                            g.sSCR[i] += abd;
                            ++nnz[i];
                        }
                    }
                    g.sSCR[i] /= nnz[i];
                }

                std::partial_sort_copy(g.sSCR.begin(), g.sSCR.end(), abd_distr.begin(), abd_distr.end());

                rank(g.sTNF, g.sTNF, "min");

                std::vector<double> sCOR;
                if (nABD >= minSample) {
                    sCOR.resize(nEdges);
#pragma omp parallel for
                    for (size_t i = 0; i < nEdges; ++i) {
                        size_t r1 = g.to[i], r2 = g.from[i];
                        sCOR[i] = cal_abd_corr(r1, r2);
                    }
                    rank(sCOR, sCOR, "max");
                }

#pragma omp parallel for
                for (size_t i = 0; i < nEdges; ++i) {
                    g.sTNF[i] = abd_distr[round(g.sTNF[i]) - 1];  // fit tnf to abd (consider abd as reference distribution)

                    if (nABD >= minSample) sCOR[i] = abd_distr[round(sCOR[i]) - 1];

                    double wTNF = 1. / (1 + nnz[i]);

                    if (nABD >= minSample)
                        g.sSCR[i] = POW(POW(g.sSCR[i], 1. - wTNF) * POW(g.sTNF[i], wTNF) * sCOR[i], 1. / 2.);  // geometric mean
                    else
                        g.sSCR[i] = POW(g.sSCR[i], 1. - wTNF) * POW(g.sTNF[i], wTNF);
                }
            } else {
                g.sSCR = g.sTNF;
            }

            std::vector<double>().swap(g.sTNF);
            ABD_VAR.clear();
            ABD_VAR.resize(0, 0, false);

            if (debug)
                cout << *std::min_element(g.sSCR.begin(), g.sSCR.end()) << " : " << *std::max_element(g.sSCR.begin(), g.sSCR.end())
                     << endl;

            // 4. build sequential graph covering x % nodes and do clustering and add more edges and do clustering again
            std::vector<size_t> oSCR;
            orderhigh(g.sSCR, oSCR);

            std::vector<size_t> node_order;

            std::vector<Similarity> p_schedule2;
            for (size_t i = 1; i <= 10; ++i) p_schedule2.push_back(maxP / 10 * i);

            mems.resize(nobs);
            std::iota(mems.begin(), mems.end(), 0);  // each is a singleton to start
            verbose_message("Traversing graph with %d nodes and %d edges\n", nobs, nEdges);
            Graph g2(nobs, true);
            size_t which_p = 0;
            for (size_t i = 0; i < nEdges; ++i) {
                size_t ii = g.to[oSCR[i]], jj = g.from[oSCR[i]];

                // 1. check if they are binned to the same cluster. if then skip generating additional edges.
                if (mems[ii] != mems[jj]) {  // || which_p < 5 allow all edges from first 5 schedule
                    if (g2.connected_nodes.find(ii) == g2.connected_nodes.end()) {
                        node_order.push_back(ii);
                        g2.connected_nodes.insert(ii);
                    }
                    if (g2.connected_nodes.find(jj) == g2.connected_nodes.end()) {
                        node_order.push_back(jj);
                        g2.connected_nodes.insert(jj);
                    }

                    Similarity scr = g.sSCR[oSCR[i]];
                    if (scr > minS) {
                        g2.sSCR.push_back(scr);
                        g2.from.push_back(jj);
                        g2.to.push_back(ii);
                        g2.incs[ii].push_back(g2.from.size() - 1);
                        g2.incs[jj].push_back(g2.from.size() - 1);
                    } else {
                        i = nEdges - 1;  // early stopping
                    }
                }

                if (g2.getEdgeCount() > 0 &&
                    ((Similarity)g2.connected_nodes.size() / g2.n >= p_schedule2[which_p] || i == nEdges - 1)) {
                    // cout << "g2.sSCR.back(): " << g2.sSCR.back() << endl;
                    label_propagation(g2, mems, node_order);
                    verbose_message(
                        "Building SCR Graph and Binning (%d vertices and %d edges) [P = %2.2f%%; %.1fGb / %.1fGb]                 "
                        "    "
                        "      \n",
                        g2.connected_nodes.size(), g2.getEdgeCount(), p_schedule2[which_p] * 100, getUsedPhysMem(),
                        getTotalPhysMem() / 1024 / 1024);

                    if (debug) {
                        std::string osfileName("cluster.log." + boost::lexical_cast<std::string>(which_p));
                        std::ofstream os(osfileName);
                        if (!os) {
                            cerr << "[Error!] Failed to write to " << osfileName << endl;
                            return 1;
                        }
                        ClassMap _cls;
                        for (size_t i = 0; i < nobs; ++i) {
                            _cls[mems[i]].push_back(i);
                        }
                        for (size_t kk = 0; kk < nobs; ++kk) {
                            if (_cls[kk].size() > 1) {
                                os << kk << " : ";
                                ContigVector& vec = _cls[kk];
                                std::sort(vec.begin(), vec.end());
                                for (auto it2 = vec.begin(); it2 != vec.end(); ++it2) {
                                    os << *it2 << ",";
                                }
                                os << endl;
                            }
                        }
                        os.close();
                        if (!os) {
                            cerr << "[Error!] Failed to write to " << osfileName << endl;
                            return 1;
                        }
                    }

                    if (++which_p == p_schedule2.size()) break;
                }
            }
        }

        for (size_t i = 0; i < nobs; ++i) {
            cls[mems[i]].push_back(i);
        }

        if (verbose) {
#pragma omp parallel for reduction(+ : totalSize)
            for (size_t i = 0; i < nobs; ++i) {
                totalSize += sizes[i];
            }
// for ƒ(auto it = seqs.begin(); it != seqs.end(); ++it) totalSize += it->size();
#pragma omp parallel for reduction(+ : totalSize1)
            for (size_t i = 0; i < nobs1; ++i) {
                totalSize1 += small_sizes[i];
            }
            // for (auto it = small_seqs.begin(); it != small_seqs.end(); ++it) totalSize1 += it->size();
        }

        // dissolve all small bins and give them another chance to be binned with large ones.
        std::vector<size_t> leftovers, toBeErased;
        for (auto it = cls.begin(); it != cls.end(); ++it) {
            size_t kk = it->first;

            size_t s = 0;

            for (auto it2 = cls[kk].begin(); it2 != cls[kk].end(); ++it2) {
                // s += seqs[*it2].size();
                s += sizes[*it2];
            }

            if (s < minClsSize) {
                leftovers.insert(leftovers.end(), cls[kk].begin(), cls[kk].end());
                std::vector<int>().swap(cls[kk]);
                toBeErased.push_back(kk);
            }
        }
        for (auto x : toBeErased) cls.erase(x);

        leftovers.shrink_to_fit();
        std::vector<size_t>().swap(toBeErased);

        // additional binning with small contigs
        if (!noAdd && nABD >= minSample && (leftovers.size() > 0 || nobs1 > 0)) {
            size_t minCS = 10;  // minimum class size for additional recruiting

            std::vector<float> rowMat(ABD.size2());
            for (size_t r = 0; r < ABD.size1(); ++r) {
                MatrixRowType rRow(ABD, r);
                std::copy(rRow.begin(), rRow.end(), rowMat.begin());
                rank(rowMat, rowMat);
                std::copy(rowMat.begin(), rowMat.end(), rRow.begin());
            }

            unsigned long long binned_size = 0;
            for (auto it = cls.begin(); it != cls.end(); ++it) {
                size_t kk = it->first;
                for (auto it2 = cls[kk].begin(); it2 != cls[kk].end(); ++it2) {
                    binned_size += seqs[*it2].size();
                }
            }

            // 1. calculate mean corr within bins
            // 2. cal mean corr from a contig to  a bin greater than mean and assign it to the best bin over the threshold
            std::unordered_map<size_t, double> cls_corr;
#pragma omp parallel
#pragma omp single
            for (auto it = cls.begin(); it != cls.end(); ++it) {
                size_t kk = it->first;
                size_t cs = it->second.size();
                if (cs >= minCS) {
#pragma omp task
                    {
                        double corr = 0;
                        const auto& c = it->second;
                        for (size_t i = 0; i < cs; ++i) {
                            for (size_t j = i + 1; j < cs; ++j) {
                                corr += cal_abd_corr(c[i], c[j]);
                            }
                        }

                        double x = corr / (cs * (cs - 1) / 2);
#pragma omp critical(CALC_MEAN_CORR)
                        cls_corr[kk] = x;
                    }
                }
            }

            verbose_message("Binning lost contigs...          \n");
            ClassMap cls_leftovers;
#pragma omp parallel for schedule(dynamic, 1)
            for (size_t l = 0; l < leftovers.size(); ++l) {
                int best_cls = -1;
                for (auto it = cls.begin(); it != cls.end(); ++it) {
                    size_t kk = it->first;
                    const auto& c = it->second;
                    size_t cs = c.size();
                    if (cs >= minCS) {
                        double corr = 0;
                        size_t i = 0;

                        // subset
                        for (; i < minCS; ++i) corr += cal_abd_corr(c[i], leftovers[l]);

                        // early stop
                        if (corr / minCS < cls_corr[kk]) continue;

                        for (; i < cs; ++i) corr += cal_abd_corr(c[i], leftovers[l]);

                        corr /= cs;
                        if (corr >= cls_corr[kk]) {
                            if (best_cls > -1) {  // only allow unique assignment.
                                best_cls = -1;
                                break;
                            }
                            best_cls = kk;
                        }
                    }
                }
                if (best_cls > -1) {
#pragma omp critical(ADD_LEFTOVER_CONTIGS)
                    cls_leftovers[best_cls].push_back(leftovers[l]);
                }
            }

            verbose_message("Binning small contigs...          \n");
            ClassMap cls_small;
            if (nobs1 > 0) {
                // Spearman corr
                for (size_t r = 0; r < small_ABD.size1(); ++r) {
                    MatrixRowType rRow(small_ABD, r);
                    std::copy(rRow.begin(), rRow.end(), rowMat.begin());
                    rank(rowMat, rowMat);
                    std::copy(rowMat.begin(), rowMat.end(), rRow.begin());
                }

#pragma omp parallel for schedule(dynamic)
                for (size_t s = 0; s < nobs1; ++s) {
                    int best_cls = -1;
                    for (auto it = cls.begin(); it != cls.end(); ++it) {
                        size_t kk = it->first;
                        const auto& c = it->second;
                        size_t cs = c.size();
                        if (cs >= minCS) {
                            double corr = 0;
                            size_t i = 0;
                            // subset
                            for (; i < minCS; ++i) corr += cal_abd_corr(c[i], s, true);

                            // early stop
                            if (corr / minCS < cls_corr[kk]) continue;

                            for (; i < cs; ++i) corr += cal_abd_corr(c[i], s, true);

                            corr /= cs;
                            if (corr >= cls_corr[kk]) {
                                if (best_cls > -1) {  // only allow unique assignment.
                                    best_cls = -1;
                                    break;
                                }
                                best_cls = kk;
                            }
                        }
                    }
                    if (best_cls > -1) {
#pragma omp critical(ADD_SMALL_CONTIGS)
                        cls_small[best_cls].push_back(s + nobs);
                    }
                }
            }

            unsigned long long added_sum = 0;
            for (auto it = cls_leftovers.begin(); it != cls_leftovers.end(); ++it) {
                size_t kk = it->first;
                for (auto it2 = cls_leftovers[kk].begin(); it2 != cls_leftovers[kk].end(); ++it2) {
                    added_sum += sizes[*it2];
                    // added_sum += seqs[*it2].size();
                }
            }

            if (added_sum > 0) {
                if ((double)added_sum / binned_size < .10) {  // allow only at most 10% recruiting
                    for (auto it = cls_leftovers.begin(); it != cls_leftovers.end(); ++it) {
                        size_t kk = it->first;
                        cls[kk].insert(cls[kk].end(), cls_leftovers[kk].begin(), cls_leftovers[kk].end());
                    }
                    verbose_message("%2.2f%% (%lld bases) of large (>=%d) contigs were re-binned out of small bins (<%d).\n",
                                    (double)added_sum / binned_size * 100, added_sum, minContig, minClsSize);
                } else {
                    verbose_message(
                        "[Info] Additional binning of lost contigs was ignored since it was too excessive [%2.2f%% (%lld bases) "
                        "of "
                        "large (>=%d) contigs is > %2.0f%%].\n",
                        (double)added_sum / binned_size * 100, added_sum, minContig, .10 * 100);
                }
            }

            added_sum = 0;
            for (auto it = cls_small.begin(); it != cls_small.end(); ++it) {
                size_t kk = it->first;
                for (auto it2 = cls_small[kk].begin(); it2 != cls_small[kk].end(); ++it2) {
                    added_sum += small_sizes[*it2 - nobs];
                    // added_sum += small_seqs[*it2 - nobs].size();
                }
            }

            if (added_sum > 0) {
                if ((double)added_sum / totalSize1 < .15) {  // allow only at most 15% recruiting
                    for (auto it = cls_small.begin(); it != cls_small.end(); ++it) {
                        size_t kk = it->first;
                        cls[kk].insert(cls[kk].end(), cls_small[kk].begin(), cls_small[kk].end());
                    }
                } else {
                    verbose_message(
                        "[Info] Additional binning of small contigs was ignored since it was too excessive [%2.2f%% (%lld bases) "
                        "of "
                        "small (<%d) contigs is > %2.0f%%].\n",
                        (double)added_sum / totalSize1 * 100, added_sum, minContig, .10 * 100);
                }
            }
        }

        ABD.clear();
        ABD.resize(0, 0, false);
        small_ABD.clear();
        small_ABD.resize(0, 0, false);
    } while (false);

    verbose_message("Rescuing singleton large contigs\n");
    rescue_singletons(cls);

    verbose_message("Outputting bins\n");
    output_bins(cls);
    verbose_message("Finished\n");
    hipHostFree(_mem);
    hipFree(TNF_d);
    hipFree(contig_log);
    return 0;
}