#include "hip/hip_runtime.h"
#include "metabat1.h"

int main(int ac, char* av[]) {
	std::string saveTNFFile, saveDistanceFile;
	po::options_description desc("Allowed options", 110, 110/2);
	desc.add_options()
			("help,h", "produce help message")
			("inFile,i", po::value<std::string>(&inFile), "Contigs in fasta file format [Mandatory]")
			("outFile,o", po::value<std::string>(&outFile), "Base file name for each bin. The default output is fasta format. Use -l option to output only contig names [Mandatory]")
			("abdFile,a", po::value<std::string>(&abdFile), "A file having mean and variance of base coverage depth (tab delimited; the first column should be contig names, and the first row will be considered as the header and be skipped) [Optional]")
			("cvExt", po::value<bool>(&cvExt)->zero_tokens(), "When a coverage file without variance (from third party tools) is used instead of abdFile from jgi_summarize_bam_contig_depths")
			("pairFile,p", po::value<std::string>(&pairFile), "A file having paired reads mapping information. Use it to increase sensitivity. (tab delimited; should have 3 columns of contig index (ordered by), its mate contig index, and supporting mean read coverage. The first row will be considered as the header and be skipped) [Optional]")
			("p1", po::value<Similarity>(&p1)->default_value(0), "Probability cutoff for bin seeding. It mainly controls the number of potential bins and their specificity. The higher, the more (specific) bins would be. (Percentage; Should be between 0 and 100)")
			("p2", po::value<Similarity>(&p2)->default_value(0), "Probability cutoff for secondary neighbors. It supports p1 and better be close to p1. (Percentage; Should be between 0 and 100)")
			("minProb", po::value<Similarity>(&minProb)->default_value(0), "Minimum probability for binning consideration. It controls sensitivity. Usually it should be >= 75. (Percentage; Should be between 0 and 100)")
			("minBinned", po::value<Similarity>(&minBinned)->default_value(0), "Minimum proportion of already binned neighbors for one's membership inference. It contorls specificity. Usually it would be <= 50 (Percentage; Should be between 0 and 100)")
			("verysensitive", po::value<bool>(&verysensitive)->zero_tokens(), "For greater sensitivity, especially in a simple community. It is the shortcut for --p1 90 --p2 85 --pB 20 --minProb 75 --minBinned 20 --minCorr 90")
			("sensitive", po::value<bool>(&sensitive)->zero_tokens(), "For better sensitivity [default]. It is the shortcut for --p1 90 --p2 90 --pB 20 --minProb 80 --minBinned 40 --minCorr 92")
			("specific", po::value<bool>(&specific)->zero_tokens(), "For better specificity. Different from --sensitive when using correlation binning or ensemble binning. It is the shortcut for --p1 90 --p2 90 --pB 30 --minProb 80 --minBinned 40 --minCorr 96")
			("veryspecific", po::value<bool>(&veryspecific)->zero_tokens(), "For greater specificity. No correlation binning for short contig recruiting. It is the shortcut for --p1 90 --p2 90 --pB 40 --minProb 80 --minBinned 40")
			("superspecific", po::value<bool>(&superspecific)->zero_tokens(), "For the best specificity. It is the shortcut for --p1 95 --p2 90 --pB 50 --minProb 80 --minBinned 20")
			("minCorr", po::value<Distance>(&minCorr)->default_value(0), "Minimum pearson correlation coefficient for binning missed contigs to increase sensitivity (Helpful when there are many samples). Should be very high (>=90) to reduce contamination. (Percentage; Should be between 0 and 100; 0 disables)")
			("minSamples", po::value<size_t>(&minSamples)->default_value(10), "Minimum number of sample sizes for considering correlation based recruiting")
			("minCV,x", po::value<Distance>(&minCV)->default_value(1), "Minimum mean coverage of a contig to consider for abundance distance calculation in each library")
			("minCVSum", po::value<Distance>(&minCVSum)->default_value(2), "Minimum total mean coverage of a contig (sum of all libraries) to consider for abundance distance calculation")
			("minClsSize,s", po::value<size_t>(&minClsSize)->default_value(200000), "Minimum size of a bin to be considered as the output")
			("minContig,m", po::value<size_t>(&minContig)->default_value(2500), "Minimum size of a contig to be considered for binning (should be >=1500; ideally >=2500). If # of samples >= minSamples, small contigs (>=1000) will be given a chance to be recruited to existing bins by default.")
			("minContigByCorr", po::value<size_t>(&minContigByCorr)->default_value(1000), "Minimum size of a contig to be considered for recruiting by pearson correlation coefficients (activated only if # of samples >= minSamples; disabled when minContigByCorr > minContig)")
			("numThreads,t", po::value<size_t>(&numThreads)->default_value(0), "Number of threads to use (0: use all cores)")
			("minShared", po::value<Similarity>(&minShared)->default_value(50), "Percentage cutoff for merging fuzzy contigs")
			("fuzzy", po::value<bool>(&fuzzy)->zero_tokens(), "Binning with fuzziness which assigns multiple memberships of a contig to bins (activated only with --pairFile at the moment)")
			("onlyLabel,l", po::value<bool>(&onlyLabel)->zero_tokens(), "Output only sequence labels as a list in a column without sequences")
			("sumLowCV,S", po::value<bool>(&sumLowCV)->zero_tokens(), "If set, then every sample that falls below the minCV will be used in an aggregate sample")
			("maxVarRatio,V", po::value<Distance>(&maxVarRatio)->default_value(maxVarRatio), "Ignore any contigs where variance / mean exceeds this ratio (0 disables)")
			("saveTNF", po::value<std::string>(&saveTNFFile), "File to save (or load if exists) TNF matrix for each contig in input")
			("saveDistance", po::value<std::string>(&saveDistanceFile), "File to save (or load if exists) distance graph at lowest probability cutoff")
			("saveCls", po::value<bool>(&saveCls)->zero_tokens(), "Save cluster memberships as a matrix format")
			("unbinned", po::value<bool>(&outUnbinned)->zero_tokens(), "Generate [outFile].unbinned.fa file for unbinned contigs")
			("noBinOut", po::value<bool>(&noBinOut)->zero_tokens(), "No bin output. Usually combined with --saveCls to check only contig memberships")
			("B,B", po::value<int>(&B)->default_value(20), "Number of bootstrapping for ensemble binning (Recommended to be >=20)")
			("pB", po::value<double>(&pB)->default_value(50), "Proportion of shared membership in bootstrapping. Major control for sensitivity/specificity. The higher, the specific. (Percentage; Should be between 0 and 100)")
			("seed", po::value<unsigned long long>(&seed)->default_value(0), "For reproducibility in ensemble binning, though it might produce slightly different results. (0: use random seed)")
			("keep", po::value<bool>(&keep)->zero_tokens(), "Keep the intermediate files for later usage")
			("debug,d", po::value<bool>(&debug)->zero_tokens(), "Debug output")
			("verbose,v", po::value<bool>(&verbose)->zero_tokens(), "Verbose output");

	po::variables_map vm;
	po::store(po::command_line_parser(ac, av).options(desc).allow_unregistered().run(), vm);
	po::notify(vm);

	if (vm.count("help") || inFile.length() == 0 || outFile.length() == 0) {
		cerr << "\nMetaBAT: Metagenome Binning based on Abundance and Tetranucleotide frequency (version 1:" << version << "; " << DATE << ")" << endl;
		cerr << "by Don Kang (ddkang@lbl.gov), Jeff Froula, Rob Egan, and Zhong Wang (zhongwang@lbl.gov) \n" << endl;
		cerr << desc << endl << endl;

		if(!vm.count("help")) {
			if (inFile.empty()) {
				cerr << "[Error!] There was no --inFile specified" << endl;
			}
			if (outFile.empty()) {
				cerr << "[Error!] There was no --outFile specified" << endl;
			}

		}

		return vm.count("help") ? 0 : 1;
	}

	if(verbose)
		gettimeofday(&t1, NULL);

	if (seed == 0)
		seed = time(0);
	srand(seed);

	if(p1 == 0 && p2 == 0) {
		int labeledOpts = (verysensitive ? 1 : 0) + (sensitive ? 1 : 0) + (specific ? 1 : 0) + (veryspecific ? 1 : 0) + (superspecific ? 1 : 0);
		if (labeledOpts > 1) {
			cerr << "[Error!] Please only specify one of the following options: " << endl << "\t--verysensitive, --sensitive, --specific or --veryspecific or --superspecific" << endl;
			return 1;
		}
		if (labeledOpts == 0)
			sensitive = true; // set the default, if none were specified

		if(verysensitive) {
			p1 = 90; p2 = 85; minProb = 75; minBinned = 30; minCorr = 90; p3 = 90; //pB = pB ? pB : 20;
		} else if(sensitive) {
			p1 = 90; p2 = 90; minProb = 80; minBinned = 40; minCorr = 92;
		} else if(specific) {
			p1 = 90; p2 = 90; minProb = 80; minBinned = 40; minCorr = 96;
		} else if(veryspecific) {
			p1 = 90; p2 = 90; minProb = 80; minBinned = 40;
		} else if(superspecific) {
			p1 = 95; p2 = 90; minProb = 80; minBinned = 20;
		}
	}

	if(minContig < 1500) {
		cerr << "[Error!] Contig length < 1500 is not allowed to be used for binning, rather use smaller minContigByCorr value to achieve better sensitivity" << endl;
		return 1;
	}

	if(minContigByCorr > minContig) { //disabling correlation based recruiting
		minCorr = 0;
	}

	if(minClsSize < seedClsSize) {
		cerr << "[Error!] minClsSize should be >= " << seedClsSize << endl;
		return 1;
	}

	if(p1 <= 0 || p1 >= 100) {
		cerr << "[Error!] p1 should be greater than 0 and less than 100" << endl;
		return 1;
	}

	if(p2 <= 0 || p2 >= 100) {
		cerr << "[Error!] p2 should be greater than 0 and less than 100" << endl;
		return 1;
	}

	if(p3 <= 0 || p3 >= 100) {
		cerr << "[Error!] p3 should be greater than 0 and less than 100" << endl;
		return 1;
	}

	if(pB < 0 || pB > 100) {
		cerr << "[Error!] pB should be >= 0 and <= 100" << endl;
		return 1;
	}

	if(minProb <= 0 || minProb >= 100) {
		cerr << "[Error!] minProb should be greater than 0 and less than 100" << endl;
		return 1;
	}

	if(minBinned <= 0 || minBinned >= 100) {
		cerr << "[Error!] minBinned should be greater than 0 and less than 100" << endl;
		return 1;
	}

	if(minShared < 0 || minShared > 100) {
		cerr << "[Error!] minShared should be >= 0 and <= 100" << endl;
		return 1;
	}

	if(minCV < 0) {
		cerr << "[Error!] minCV should be non-negative" << endl;
		return 1;
	}

	if (B <= 1) {
		B = 1;
		useEB = false;
	}

	if (useEB) {
		if (B < 10)
			cerr << "[Warning!] B < 10 may not be effective for ensemble binning. Consider B >= 20" << endl;
	}

	gen_commandline_hash();

	p1 /= 100.; p2 /= 100.; p3 /= 100.; pB /= 100.; minProb /= 100.; minBinned /= 100.; minShared /= 100.;

	boost::filesystem::path dir(outFile);
	if (dir.parent_path().string().length() > 0) {
		if (boost::filesystem::is_regular_file(dir.parent_path())) {
			cerr << "Cannot create directory: " << dir.parent_path().string() << ", which exists as a regular file." << endl;
			return 1;
		}
		boost::filesystem::create_directory(dir.parent_path());
	}

	print_message("MetaBAT 1 (%s) using p1 %2.1f%%, p2 %2.1f%%, p3 %2.1f%%, minProb %2.1f%%, minBinned %2.0f%%, minCV %2.1f, "
			"minContig %d, minContigByCorr %d, minCorr %2.0f%%, paired %d, and %d bootstrapping\n",
			version.c_str(), p1*100, p2*100, p3*100, minProb*100, minBinned*100, minCV, minContig,
			minContigByCorr, minCorr, pairFile.length() > 0, useEB ? B : 0);

	if(numThreads == 0)
		numThreads = omp_get_max_threads();
	else
		numThreads = std::min(numThreads, (size_t) omp_get_max_threads());
	omp_set_num_threads(numThreads);

	//std::ifstream is;

	for(size_t i = 0; i < nTNF; ++i) {
		TNmap[TN[i]] = i;
	}

	for(size_t i = 0; i < 16; ++i) {
		TNPmap.insert(TNP[i]);
	}

	nobs = 0;
	int nresv = 0; //number of reserved contigs, usually between minContigByCorrForGraph and minContigByCorr

	FILE *fp = fopen(inFile.c_str(), "r");
	if (fp == NULL) {
		cerr << "[Error!] can't open the sequence fasta file " << inFile << endl;
		return 1;
	} else {
		fseek(fp, 0L, SEEK_END);
        fsize = ftell(fp);  // obtener el tamaño del archivo
        fclose(fp);
		hipHostMalloc((void **)&_mem, fsize);

        int fpint = open(inFile.c_str(), O_RDWR | O_CREAT, S_IREAD | S_IWRITE | S_IRGRP | S_IROTH);
        std::thread readerThreads[numThreads];

        for (int i = 0; i < numThreads; i++) {
            size_t _size;
            if (i != numThreads - 1)
                _size = chunk;
            else
                _size = chunk + (fsize % numThreads);
            readerThreads[i] = std::thread(reader, fpint, i, chunk, _size, _mem);
        }
		for (int i = 0; i < numThreads; i++) {  // esperar a que terminen de leer
            readerThreads[i].join();
        }
		close(fpint);

		size_t __min = std::min(minContigByCorr, minContigByCorrForGraph);

		size_t contig_name_i;
        size_t contig_name_e;
        size_t contig_i;
        size_t contig_e;
        size_t contig_size;
		seqs.reserve(fsize % __min);
        ignored.reserve(fsize % __min);
        contig_names.reserve(fsize % __min);
        lCtgIdx.reserve(fsize % __min);
        gCtgIdx.reserve(fsize % __min);
        //seqs_h_index_i.reserve(fsize % __min);
        //seqs_h_index_e.reserve(fsize % __min);
        for (size_t i = 0; i < fsize; i++) {  // leer el archivo caracter por caracter
            if (_mem[i] < 65) {
                contig_name_i = i;  // guardar el inicio del nombre del contig
                while (_mem[i] != 10) i++;
                contig_name_e = i;  // guardar el final del nombre del contig
                i++;
                contig_i = i;  // guardar el inicio del contig
                while (i < fsize && _mem[i] != 10) i++;
                contig_e = i;  // guardar el final del contig
                contig_size = contig_e - contig_i;
                if (contig_size >= __min) {
                    if (contig_size < minContig) {
                        if (contig_size >= minContigByCorr)
                            smallCtgs.insert(nobs);
                        else
                            nresv++;
                    }
                    //seqs_h_index_i.emplace_back(contig_i);
                    //seqs_h_index_e.emplace_back(contig_e);
                    lCtgIdx[std::string_view(_mem + contig_name_i, contig_name_e - contig_name_i)] = nobs;
                    gCtgIdx[nobs++] = seqs.size();
                } else {
                    ignored[std::string_view(_mem + contig_name_i, contig_name_e - contig_name_i)] = seqs.size();
                }
                contig_names.emplace_back(std::string_view(_mem + contig_name_i, contig_name_e - contig_name_i));
                seqs.emplace_back(std::string_view(_mem + contig_i, contig_e - contig_i));
            }
        }
        //seqs_h_index_i.shrink_to_fit();  // liberar memoria no usada
        //seqs_h_index_e.shrink_to_fit();  // liberar memoria no usada
        seqs.shrink_to_fit();            // liberar memoria no usada
        contig_names.shrink_to_fit();    // liberar memoria no usada
	}

	assert(nobs == lCtgIdx.size());

	nobs2 = ignored.size();

	verbose_message("Finished reading %d contigs. Number of target contigs >= %d are %d, and [%d and %d) are %d \n", nobs + nobs2, minContig, nobs - smallCtgs.size() - nresv, minContigByCorr, minContig, smallCtgs.size());

	if(contig_names.size() != nobs + nobs2 || seqs.size() != nobs + nobs2) {
		cerr << "[Error!] Need to check whether there are duplicated sequence ids in the assembly file" << endl;
		return 1;
	}

	nresv = 0;

	const int nNonFeat = cvExt ? 1 : 3; //number of non features

	if(abdFile.length() > 0) {
		smallCtgs.clear();
		std::unordered_map<std::string, size_t> lCtgIdx2;
		std::unordered_map<size_t, size_t> gCtgIdx2;

		nobs = std::min(nobs, countLines(abdFile.c_str()) - 1); //the first row is header
		if (nobs < 1) {
			cerr << "[Error!] There are no lines in the abundance depth file or fasta file!" << endl;
			exit(1);
		}
		nABD = ncols(abdFile.c_str(), 1) - nNonFeat; //num of features (excluding the first three columns which is the contigName, contigLen, and totalAvgDepth);
		if(!cvExt) {
			if(nABD % 2 != 0) {
				cerr << "[Error!] Number of columns (excluding the first column) in abundance data file is not even." << endl;
				return 1;
			}
			nABD /= 2;
		}
		ABD.resize(nobs, nABD);
		ABD_VAR.resize(nobs, nABD);

		std::ifstream is(abdFile.c_str());
		if (!is.is_open()) {
			cerr << "[Error!] can't open the contig coverage depth file " << abdFile << endl;
			return 1;
		}

		int r = -1;
		int nskip = 0;

		for (std::string row; safeGetline(is, row) && is.good(); ++r) {
			if(r == -1) //the first row is header
				continue;

			std::stringstream ss(row);
			int c = -nNonFeat;
			Distance mean, variance, meanSum = 0;
			std::string label;
			bool isGood = true;
			DistancePair tmp(0,0);

			for (std::string col; getline(ss, col, tab_delim); ++c) {
				if (col.empty())
					break;
				if (c == -3 || (cvExt && c == -1)) { //contig name
					trim_fasta_label(col);
					label = col;
					if(lCtgIdx.find(label) == lCtgIdx.end()) {
						if(ignored.find(label) == ignored.end()) {
							verbose_message("[Warning!] Cannot find the contig (%s) in abundance file from the assembly file\n", label.c_str());
						} else if(debug) {
							verbose_message("[Info] Ignored a small contig (%s) having length %d < %d\n", label.c_str(), seqs[ignored[label]].size(), minContig);
						}
						isGood = false; //cannot find the contig from fasta file. just skip it!
						break;
					}
					continue;
				} else if (c == -2) {
					continue;
				} else if (c == -1) {
					meanSum = boost::lexical_cast<Distance>(col.c_str());
					if (meanSum < minCVSum) {
						if(debug)
							verbose_message("[Info] Ignored a contig (%s) having mean coverage %2.2f < %2.2f \n", label.c_str(), meanSum, minCVSum);
						isGood = false; //cannot find the contig from fasta file. just skip it!
						break;
					}
					continue;
				}

				assert(r - nskip >= 0 && r - nskip < (int) nobs);

				bool checkMean = false, checkVar = false;

				if(cvExt) {
					mean = ABD(r - nskip, c) = boost::lexical_cast<Distance>(col.c_str());
					meanSum += mean;
					variance = ABD_VAR(r - nskip, c) = mean;
					checkMean = true;
				} else {
					if(c % 2 == 0) {
						mean = ABD(r - nskip, c/2) = boost::lexical_cast<Distance>(col.c_str());
						checkMean = true;
					} else {
						variance = ABD_VAR(r - nskip, c/2) = boost::lexical_cast<Distance>(col.c_str());
						checkVar = true;
					}
				}

				if(checkMean) {
					if(mean > 1e+7) {
						cerr << "[Error!] Need to check where the average depth is greater than 1e+7 for the contig " << label << ", column " << c + 1 << endl;
						return 1;
					}
					if(mean < 0) {
						cerr << "[Error!] Negative coverage depth is not allowed for the contig " << label << ", column " << c + 1 << ": " << mean << endl;
						return 1;
					}
				}

				if(checkVar) {
					if(variance > 1e+14) {
						cerr << "[Error!] Need to check where the depth variance is greater than 1e+14 for the contig " << label << ", column " << c + 1 << endl;
						return 1;
					}
					if(variance < 0) {
						cerr << "[Error!] Negative variance is not allowed for the contig " << label << ", column " << c + 1 << ": " << variance << endl;
						return 1;
					}
					if (maxVarRatio > 0.0 && mean > 0 && variance / mean > maxVarRatio) {
						cerr << "[Warning!] Skipping contig due to >maxVarRatio variance: " << variance << " / " << mean << " = " << variance / mean << ": " << label << endl;
						isGood = false;
						break;
					}
				}

				if(c == (int)(nABD * (cvExt ? 1 : 2) - 1)) {
					if (meanSum < minCVSum) {
						if(debug)
							verbose_message("[Info] Ignored a contig (%s) having mean coverage %2.2f < %2.2f \n", label.c_str(), meanSum, minCVSum);
						isGood = false; //cannot find the contig from fasta file. just skip it!
						break;
					}
					tmp.second = meanSum; //useEB ? rand() : meanSum
				}
			}

			if (isGood) {
				size_t _gidx = gCtgIdx[lCtgIdx[label]];
				if(seqs[_gidx].size() < minContig) {
					smallCtgs.insert(r - nskip);
					if (seqs[_gidx].size() < minContigByCorr)
						++nresv;
				}
				lCtgIdx2[label] = r - nskip; //local index
				gCtgIdx2[r - nskip] = _gidx; //global index
			} else {
				++nskip;
				continue;
			}

			tmp.first = r - nskip;
			rABD.push_back(tmp);

			if ((int) nABD != (cvExt ? c : c/2)) {
				cerr << "[Error!] Different number of variables for the object for the contig " << label << endl;
				return 1;
			}
		}
		is.close();

		verbose_message("Finished reading %d contigs (using %d including %d short contigs) and %d coverages from %s\n", r, r - nskip - nresv, smallCtgs.size() - nresv, nABD, abdFile.c_str());

		if ((specific || veryspecific) && nABD < minSamples) {
			cerr << "[Warning!] Consider --superspecific for better specificity since both --specific and --veryspecific would be the same as --sensitive when # of samples (" << nABD << ") < minSamples (" << minSamples << ")" << endl;
		}

		if (nABD < minSamples) {
			cerr << "[Info] Correlation binning won't be applied since the number of samples (" << nABD << ") < minSamples (" << minSamples << ")" << endl;
		}

		for(std::unordered_map<std::string, size_t>::const_iterator it = lCtgIdx.begin(); it != lCtgIdx.end(); ++it) {
			if(lCtgIdx2.find(it->first) == lCtgIdx2.end()) { //given seq but missed depth info or skipped
				ignored[it->first] = gCtgIdx[it->second];
			}
		}

		lCtgIdx.clear();
		gCtgIdx.clear();

		lCtgIdx = lCtgIdx2;
		gCtgIdx = gCtgIdx2;

		assert(lCtgIdx.size() == gCtgIdx.size());
		assert(lCtgIdx.size() + ignored.size() == seqs.size());

		nobs = lCtgIdx.size();
		nobs2 = ignored.size();

		if(ABD.size1() != nobs) {
			ABD.resize(nobs, nABD, true);
			ABD_VAR.resize(nobs, nABD, true);
		}

		assert(rABD.size() == nobs);
	}

	if (!loadTNFFromFile(saveTNFFile, minContig)) {
		ProgressTracker progress(nobs);

		// cambia el tamaño de la matriz TNF a nobs (numero de contigs) * nTNF (cantidad de combinaciones de 4 bases ¿136?)
		TNF.resize(nobs, nTNF);
		TNF.clear();

		//paralelo
		#pragma omp parallel for schedule (dynamic)
		for (size_t r = 0; r < nobs; ++r)
		{
			// omite el contig si pertenece a los smallcontigs

			if(smallCtgs.find(r) == smallCtgs.end()) { //TNF is meaningless for small contigs
				// (solo funciona de debug) error si se procesa un contig almacenado en el set de ignorados
				assert(ignored.find(contig_names[gCtgIdx[r]]) == ignored.end());

				//obtiene la secuencia del contig
				//-------------- necesito guardar seqs en la memoria de GPU
				std::string& s = seqs[gCtgIdx[r]];

				// crea la variable tn para almacenar las 4 bases
				char tn[5] = {'\0'};

				for (size_t i = 0; i < s.length() - 3; ++i) {
					// copia desde s a tn 4 bases partiendo desde i 
					s.copy(tn, 4, i);

					// busca la secuencia encontrada en TNmap (para obtener el identificador de la secuencia)
					//---------------- necesito almacenat el mapa TNmap en GPU
					std::unordered_map<std::string, int>::iterator it = TNmap.find(tn);

					// si no es un palindromo aumenta el contador en la posición r, identificador de tn
					if(it != TNmap.end())
						++TNF(r, it->second);

					//********** aquí debería haber un continue si la condición de arriba no se cumple

					//reverse complement
					// obtiene el reverso de tn
					std::reverse(tn, tn+4);
					// modifica tn para obtener el complemento, error si el string no es correcto
					if(!revComp(tn,4)) {
						//cout << "Unknown nucleotide letter found: " << s.substr(i, 4) << " in the row " << r + 1 << endl;
						continue;
					}

					// si no es un palindromo aumenta el contador en la posición r, identificador de tn(complemento)
					if (TNPmap.find(tn) == TNPmap.end()) { //if it is palindromic, then skip
						it = TNmap.find(tn);
						if(it != TNmap.end()) //********************** consulta innecesaria
							++TNF(r, it->second);
					}
				}

				
				//normalize to unit size (L2 norm)
				Distance rsum = 0;
				for(size_t c = 0; c < TNF.size2(); ++c) {
					rsum += TNF(r,c) * TNF(r,c);
				}
				rsum = SQRT(rsum);
				for(size_t c = 0; c < TNF.size2(); ++c) {
					TNF(r,c) /= rsum;
				}
			}

			//texto de progreso
			if(verbose) {
				progress.track();
				if (omp_get_thread_num() == 0 && progress.isStepMarker()) {
					verbose_message("Calculating TNF %s\r", progress.getProgress());
				}
			}
		}
		// almacena la matriz de TNF
		saveTNFToFile(saveTNFFile, minContig);
	}
	verbose_message("Finished TNF calculation.                                  \n");

	//need to build rABD when no ABD available.
	if(rABD.size() == 0) {
		for(size_t i = 0; i < nobs; ++i) {
			rABD.push_back(std::make_pair(i, rand()));
		}
	}

	// find the upper bound for distances to fill the sparse matrix
	Distance requiredMinP = std::min(std::min(std::min(p1, p2), p3), minProb);
	if (requiredMinP > .75) //allow every mode exploration without reforming graph.
		requiredMinP = .75;

	if (!loadDistanceFromFile(saveDistanceFile, requiredMinP, minContig)) {
		ProgressTracker progress = ProgressTracker(nobs * (nobs-1) / 2, nobs / 100 + 1);

		gprob.m_vertices.resize(nobs);

		#pragma omp parallel for schedule (dynamic)
		for (size_t i = 1; i < nobs; ++i) {
			if(smallCtgs.find(i) == smallCtgs.end()) { //Don't build graph for small contigs
				for (size_t j = 0; j < i; ++j) { // populate lower triangle
					if(smallCtgs.find(j) != smallCtgs.end()) //Don't build graph for small contigs
						continue;
					bool passed = false;
					Similarity s = 1. - cal_dist(i, j, 1. - requiredMinP, passed);
					if (passed && s >= requiredMinP) {
						#pragma omp critical (ADD_EDGE_1)
						{
							boost::add_edge(i, j, Weight(s), gprob);
						}
					}
				}
			}

			if (verbose) {
				progress.track(i);
				if(omp_get_thread_num() == 0 && progress.isStepMarker())
					verbose_message("Building a probabilistic graph: %s\r", progress.getProgress());
			}
		}

		saveDistanceToFile(saveDistanceFile, requiredMinP, minContig);
	}

	verbose_message("Finished building a probabilistic graph. (%d vertices and %d edges)          \n", boost::num_vertices(gprob), boost::num_edges(gprob));

	gIdx = boost::get(boost::vertex_index, gprob);
	gWgt = boost::get(boost::edge_weight, gprob);

	bool good_pair = pairFile.length() > 0 && readPairFile();

	boost::numeric::ublas::matrix<size_t> resES(nobs, B, 0);

	ClassMap cls;

	if (!loadBootFromFile(resES)) {
		for (int b = 0; b < B; ++b) {
			ContigVector _medoid_ids;
			std::vector<double> medoid_vals;
			ContigSet binned;
			ContigSet leftovers;
			ClassIdType good_class_ids;
			cls.clear();

			if (b > 0) {
				if (rABD.size() > 0) {
					for(std::list< DistancePair >::iterator it = rABD.begin(); it != rABD.end(); ++it) {
						it->second = rand();
						rABD2.push_back(*it);
					}
					rABD.clear();
				}
				rABD = rABD2;
				rABD2.clear();
			}
			rABD.sort(cmp_abd);

			pam(_medoid_ids, medoid_vals, binned, cls, leftovers, good_class_ids);

			if(!useEB)
				verbose_message("Leftover contigs before fish_more: %2.2f%% (%d out of %d)\n", (double) leftovers.size() / nobs * 100., leftovers.size(), nobs);

			bool leftout = true;
			int fished = 1;
			while (leftout) {
				leftout = false;

				fish_more_by_friends_membership(cls, leftovers, good_class_ids);
				if(!useEB)
					verbose_message("Leftover contigs after fish_more_by_friends_membership (roughly): %2.2f%% (%d out of %d), %d bins   \r", (double) leftovers.size() / nobs * 100., leftovers.size(), nobs, good_class_ids.size());

				ClassIdType good_class_ids2;
				for(ClassIdType::const_iterator it = good_class_ids.begin(); it != good_class_ids.end(); ++it) {
					size_t s = 0;
					size_t kk = *it;
					for(ContigVector::iterator it2 = cls[kk].begin(); it2 != cls[kk].end(); ++it2) {
						s += seqs[gCtgIdx[*it2]].size();
					}
					if (s < std::min(seedClsSize * (size_t) std::pow(2,fished), minClsSize)) {
						leftovers.insert(cls[kk].begin(), cls[kk].end());
						leftout = true;
					} else
						good_class_ids2.insert(kk);
				}

				good_class_ids = good_class_ids2;

				fished++;
			}

			if(!useEB)
				cout << endl;

			for (ClassIdType::const_iterator it = good_class_ids.begin(); it != good_class_ids.end(); ++it) {
				fish_more(*it, cls, leftovers);
			}
			if(!useEB)
				verbose_message("Leftover contigs after fish_more (roughly): %2.2f%% (%d out of %d)\n", (double) leftovers.size() / nobs * 100., leftovers.size(), nobs);

			if(minCorr > 0) {
				size_t fished = fish_more_by_corr(_medoid_ids, cls, leftovers, good_class_ids);
				if(!useEB)
					verbose_message("Leftover contigs after fish_more_by_corr (roughly): %2.2f%% (%d out of %d)\n", (double) (leftovers.size()-fished) / nobs * 100., (leftovers.size()-fished), nobs);
			}

			if(good_pair) {
				fish_pairs(binned, cls, good_class_ids);

				if(!useEB) {
					verbose_message("Number of clusters formed before merging: %d\n", good_class_ids.size()); //# of bins >= 2 members
					verbose_message("Merging bins that share >= %2.2f%%\n", minShared * 100.);
				}
				//sort bin by # of contigs; for each bin; find the first bin that shared >= minShared and merge two bins; iterate
				size_t k = 0;

				//convert cls => cls bit set where each element represent each contig
				std::unordered_map<int, boost::dynamic_bitset<> > clsB;
				for(ClassIdType::const_iterator it = good_class_ids.begin(); it != good_class_ids.end(); ++it) {
					boost::dynamic_bitset<> bs(seqs.size());
					#pragma omp parallel for
					for(size_t m = 0; m < cls[*it].size(); ++m) {
						#pragma omp critical (FUZZY_1)
						bs[cls[*it][m]] = 1;
					}
					assert(bs.count() == cls[*it].size());
					clsB[*it] = bs;
					assert(bs.count() == clsB[*it].count());
				}

				while(k < good_class_ids.size()) {
					std::vector<ClsSizePair> cls_size;
					for(ClassIdType::const_iterator it = good_class_ids.begin(); it != good_class_ids.end(); ++it) {
						ClsSizePair csp(*it, cls[*it].size());
						cls_size.push_back(csp);
					}
					sort(cls_size.begin(), cls_size.end(), cmp_cls_size);

					int cls1 = cls_size[k].first;

					bool isMerged = false;
					std::vector<size_t> kk_hist(omp_get_max_threads(), cls_size.size());

					#pragma omp parallel for schedule (static, 1)
					for(size_t kk = k + 1; kk < cls_size.size(); ++kk) {
						if(isMerged)
							continue;
						int cls2 = cls_size[kk].first;

						boost::dynamic_bitset<> tmp = clsB[cls2] & clsB[cls1];
						double shared = (double) tmp.count() / cls_size[k].second;

						if(debug && !useEB && omp_get_thread_num() == 0)
							verbose_message("clsB[cls2]: %d, clsB[cls1]: %d, tmp: %d, cls_size[k].second: %d, shared: %2.2f\n", clsB[cls2].count(), clsB[cls1].count(), tmp.count(), cls_size[k].second, shared * 100);

						if(shared >= minShared) {
							if(!useEB && omp_get_thread_num() == 0)
								verbose_message("Bin %d and %d were merged to %d (%2.2f%% shared)\n", cls1+1, cls2+1, cls2+1, shared * 100.);
							kk_hist[omp_get_thread_num()] = kk;
							isMerged = true;
						}
					}

					if(isMerged) {
						size_t kk = *std::min_element(kk_hist.begin(), kk_hist.end());
						k = 0; //reset whenever any bins are combined so that it start from the smallest again (inefficient but most thorough way)
						size_t cls2 = cls_size[kk].first;
						//combine cls1 and cls2 => make it as cls2
						clsB[cls2] |= clsB[cls1];
						clsB.erase(cls1);
						ContigSet tmp;
						tmp.insert(cls[cls2].begin(), cls[cls2].end());
						tmp.insert(cls[cls1].begin(), cls[cls1].end());
						cls[cls2].clear();
						cls[cls2].insert(cls[cls2].end(), tmp.begin(), tmp.end());
						cls[cls1].clear();
						cls.erase(cls1);
						good_class_ids.erase(cls1);
					} else { //k and kk were not merged
						++k;
					}

					if(debug)
						cout << "good_class_ids.size(): " << good_class_ids.size() << ", kk: " << k << endl;
				}
			}

			if	(useEB) {
				for (ClassIdType::const_iterator it = good_class_ids.begin(); it != good_class_ids.end(); ++it) {
					for (ContigVector::iterator it2 = cls[*it].begin(); it2 != cls[*it].end(); ++it2) {
						resES(*it2, b) = *it;
					}
				}
				verbose_message("Bootstrapping %d/%d [%.1fGb / %.1fGb]          \r", b+1, B, getUsedPhysMem(), getTotalPhysMem() / 1024 / 1024);
			}
		}

		if (useEB) {
			verbose_message("Bootstrapping %d/%d [%.1fGb / %.1fGb]             \n", B, B, getUsedPhysMem(), getTotalPhysMem() / 1024 / 1024);
			saveBootToFile(resES);
		}
	}

	gprob.clear();
	gprob.m_edges.resize(0);
	gprob.m_vertices.resize(0); gprob.m_vertices.shrink_to_fit();
	ABD.clear();
	ABD_VAR.clear();
	TNF.clear();
	ABD.resize(0,0,false);
	ABD_VAR.resize(0,0,false);
	TNF.resize(0,0,false);

	if (useEB) {
		igraph_t g;
		igraph_empty(&g, nobs, 0);

		igraph_weight_vector_t weights;
		igraph_vector_init(&weights, 0);

		g.incs = igraph_Calloc(g.n, igraph_edge_vector_t);
		for (node_t i = 0; i < g.n; i++) {
			igraph_vector_init(&g.incs[i], 0);
		}

		ProgressTracker progress = ProgressTracker(nobs * (nobs-1) / 2, nobs / 100 + 1);

		if(!loadENSFromFile(g, weights)) {
			edge_t reserved = (edge_t) nobs * 1000;

			igraph_vector_reserve(&weights, reserved);
			igraph_vector_reserve(&g.from, reserved);
			igraph_vector_reserve(&g.to, reserved);

			size_t cutoff = (size_t) B * pB;
			std::vector<size_t> num_binned(nobs, 0);

			#pragma omp parallel for
			for (size_t i = 0; i < nobs; ++i)
				for(int j = 0; j < B; ++j)
					num_binned[i] += resES(i,j) > 0;

			#pragma omp parallel for schedule (dynamic, 100)
			for (node_t i = 0; i < nobs; ++i) {
				if (num_binned[i] >= cutoff) {
					for (node_t j = i + 1; j < nobs; ++j) {
						if(num_binned[j] < cutoff)
							continue;

						size_t _scr = 0;
						for (int h=0; h<B; ++h)
							if(resES(i,h) > 0 && resES(j,h) > 0)
								_scr += resES(i,h) == resES(j,h);

						if (_scr >= cutoff) {
							#pragma omp critical (ENSEMBLE_ADD_WEIGHT)
							{
								igraph_vector_push_back(&weights, (float)_scr / B);
								igraph_vector_push_back(&g.from, (uint_least32_t) j);
								igraph_vector_push_back(&g.to, (uint_least32_t) i);

								igraph_vector_push_back(&g.incs[i], igraph_vector_size(&g.from) - 1);
								igraph_vector_push_back(&g.incs[j], igraph_vector_size(&g.from) - 1);
							}
						}
					}
				}
				if (verbose) {
					progress.track(nobs-i-1);
					if (omp_get_thread_num() == 0 && progress.isStepMarker()) {
						verbose_message("Building Ensemble Graph %s [%.1fGb / %.1fGb]\r", progress.getProgress(), getUsedPhysMem(), getTotalPhysMem() / 1024 / 1024);
					}
				}
			}
			verbose_message("Building Ensemble Graph %s [%.1fGb / %.1fGb]\r", progress.getProgress(), getUsedPhysMem(), getTotalPhysMem() / 1024 / 1024);

			igraph_vector_resize_min(&g.to);
			igraph_vector_resize_min(&g.from);
			igraph_vector_resize_min(&weights);

			//saveENSToFile(g, weights);
		}

		verbose_message("Finished Ensemble Graph (%lld vertices and %lld edges) [%.1fGb / %.1fGb]                          \n", igraph_vcount(&g), igraph_ecount(&g), getUsedPhysMem(), getTotalPhysMem() / 1024 / 1024);

		igraph_node_vector_t membership;
		igraph_vector_init(&membership, 0);

		igraph_rng_seed(igraph_rng_default(), seed);

		verbose_message("Starting Ensemble Binning [%.1fGb / %.1fGb]\n", getUsedPhysMem(), getTotalPhysMem() / 1024 / 1024);
		igraph_community_label_propagation(&g, &membership, &weights);
		verbose_message("Finished Ensemble Binning [%.1fGb / %.1fGb]\n", getUsedPhysMem(), getTotalPhysMem() / 1024 / 1024);

		igraph_destroy(&g);
		igraph_vector_destroy(&weights);

		if(debug) {
			std::ofstream os(outFile.c_str());
			os.rdbuf()->pubsetbuf(os_buffer, buf_size);
			for (size_t i = 0; i < nobs; ++i) {
				os << contig_names[gCtgIdx[i]] << tab_delim;
				os << VECTOR(membership)[i] << line_delim;
			}
			for(std::unordered_map<std::string_view, size_t>::const_iterator it = ignored.begin(); it != ignored.end(); ++it) {
				os << contig_names[it->second] << tab_delim << 0 << line_delim;
			}
			os.close();
		}

		cls.clear();
		for (size_t i = 0; i < nobs; ++i) {
			cls[VECTOR(membership)[i]].push_back(i);
		}

		igraph_vector_destroy(&membership);
	}

	//if everything was fine, delete intermediate files
	if (!keep && useEB) {
		std::remove(("boot." + std::to_string(commandline_hash)).c_str());
		std::remove(("ens." + std::to_string(commandline_hash)).c_str());
		verbose_message("Cleaned up intermediate files\n");
	}

	Distance binnedSize = 0;

// One of ways to make the bin ids deterministic... sort bins by their size
	std::unordered_map<size_t, size_t> cls_size;
	std::vector<DistancePair> cls_med_abd;
	for(ClassMap::const_iterator it = cls.begin(); it != cls.end(); ++it) {
		int kk = it->first;
		size_t s = 0;

		for(ContigVector::iterator it2 = cls[kk].begin(); it2 != cls[kk].end(); ++it2) {
			s += seqs[gCtgIdx[*it2]].size();
		}
		binnedSize += s;
		cls_size[kk] = s;

		DistancePair dp(kk, s);
		cls_med_abd.push_back(dp);
	}
	sort(cls_med_abd.begin(), cls_med_abd.end(), cmp_abd);

	ContigSet binned;

	size_t bin_id = 1;
	for (size_t k = 0; k < cls_med_abd.size(); ++k) {
		size_t kk = cls_med_abd[k].first;

		if(!fuzzy) {
			int s = (int) cls_size[kk];
			binnedSize -= s;
			ContigSet unique;
			for(ContigVector::iterator it2 = cls[kk].begin(); it2 != cls[kk].end(); ++it2) {
				if(binned.find(*it2) != binned.end()) { //binned already
					s -= (int) seqs[gCtgIdx[*it2]].size();
				} else {
					binned.insert(*it2);
					unique.insert(*it2);
				}
			}
			cls_size[kk] = s;
			if(cls_size[kk] < minClsSize) {
				continue;
			}
			binnedSize += cls_size[kk];
			cls[kk].clear();
			cls[kk].insert(cls[kk].end(), unique.begin(), unique.end());
		}

		if(!noBinOut) {
			std::string outFile_cls = outFile + ".";
			outFile_cls.append(boost::lexical_cast<std::string>(bin_id));
			if(!onlyLabel)
				outFile_cls.append(".fa");

			std::ofstream os(outFile_cls.c_str());
			os.rdbuf()->pubsetbuf(os_buffer, buf_size);

			for(ContigVector::iterator it2 = cls[kk].begin(); it2 != cls[kk].end(); ++it2) {
				std::string_view& label = contig_names[gCtgIdx[*it2]];
				if(onlyLabel) {
					os << label << line_delim;
				} else {
					std::string_view& seq = seqs[gCtgIdx[*it2]];
					os << fasta_delim << label << line_delim;
					for(size_t s = 0; s < seq.length(); s += 60) {
						os << seq.substr(s, 60) << line_delim;
					}
				}
			}
			os.close();

			if(debug)
				cout << "Bin " << bin_id << " (" << cls_size[kk] << " bases in " << cls[kk].size() << " contigs) was saved to: " << outFile_cls << endl;
		}

		bin_id++;
	}

	if(verbose) {
		unsigned long long totalSize = 0;
		for(std::vector<std::string_view>::iterator it = seqs.begin(); it != seqs.end(); ++it)
			totalSize += it->size();
		verbose_message("%2.2f%% (%lld out of %lld bases) was binned.\n", (double) binnedSize / totalSize * 100, (unsigned long long) binnedSize, totalSize);
	}

	cout << "Number of clusters formed: " << bin_id - 1 << std::endl;

	if(saveCls || outUnbinned) {
		#pragma omp parallel for
		for (size_t k = 0; k < cls_med_abd.size(); ++k) {
			ContigVector& clsV = cls[cls_med_abd[k].first];

			//convert to global index
			for(size_t m = 0; m < clsV.size(); ++m) {
				clsV[m] = gCtgIdx[clsV[m]];
			}
		}

		std::vector<size_t> clsMap(seqs.size(), 0);
		#pragma omp parallel for
		for(size_t k = 0; k < cls_med_abd.size(); ++k) {
			size_t kk = cls_med_abd[k].first;
			for(size_t i = 0; i < cls[kk].size(); ++i) {
				assert(cls[kk][i] < (int) clsMap.size());
				clsMap[cls[kk][i]] = k + 1;
			}
		}

		if(saveCls) {
			if(!fuzzy) {
				std::ofstream os(outFile.c_str());
				os.rdbuf()->pubsetbuf(os_buffer, buf_size);

				for(size_t i = 0; i < clsMap.size(); ++i) {
					os << contig_names[i];
					os << tab_delim << clsMap[i] << line_delim;
				}
				os.flush();
				os.close();
			} else {
				//rows as contigs and columns as bins, so wanted to represent complete memberships.
			}
		}

		if(outUnbinned) {
			std::string outFile_cls = outFile + ".";
			outFile_cls.append("unbinned");
			if(!onlyLabel)
				outFile_cls.append(".fa");

			std::ofstream os(outFile_cls.c_str());
			os.rdbuf()->pubsetbuf(os_buffer, buf_size);

			for(size_t i = 0; i < clsMap.size(); ++i) {
				if (clsMap[i] == 0) {
					if(onlyLabel) {
						os << contig_names[i] << line_delim;
					} else {
						std::string_view& seq = seqs[i];
						os << fasta_delim << contig_names[i] << line_delim;
						for(size_t s = 0; s < seq.length(); s += 60) {
							os << seq.substr(s, 60) << line_delim;
						}
					}
				}
			}
			os.flush();
			os.close();
		}

	}

	return 0;
}

void fish_pairs(ContigSet& binned, ClassMap& cls, ClassIdType& good_class_ids) {

	binned.clear();

	for (ClassIdType::const_iterator it = good_class_ids.begin(); it != good_class_ids.end(); ++it) {

		ContigVector& clsV = cls[*it];

		//convert to global index
		for(size_t m = 0; m < clsV.size(); ++m) {
			clsV[m] = gCtgIdx[clsV[m]];
			binned.insert(clsV[m]);
		}
	}

	//for each cls
	//grab any reciprocal pairs
	boost::property_map<DirectedSimpleGraph, boost::vertex_index_t>::type gsIdx = boost::get(boost::vertex_index, paired);

	ContigVector good_class_ids2(good_class_ids.begin(), good_class_ids.end());

	#pragma omp parallel for schedule (dynamic)
	for (size_t k = 0; k < good_class_ids2.size(); ++k) {

		ContigVector& clsV = cls[good_class_ids2[k]];
		ContigSet clsS(clsV.begin(), clsV.end());
		assert(clsV.size() == clsS.size());

		boost::graph_traits<DirectedSimpleGraph>::out_edge_iterator e, ee, e_end, ee_end;

		bool updated = true;

		while(updated) {
			updated = false;

			ContigSet newbies;

			//grab any reciprocal pairs
			for(size_t m = 0; m < clsV.size(); ++m) {
				size_t idx = clsV[m];
				//v = boost::vertex(clsV[m], paired);
				assert(boost::out_degree(idx, paired) <= 2);
				for (boost::tie(e, e_end) = boost::out_edges(idx, paired); e != e_end; ++e) {
					int pp = boost::get(gsIdx, boost::target(*e, paired));
					if(binned.find(pp) != binned.end()) //don't recruit already binned contigs
						continue;
					if(clsS.find(pp) == clsS.end()) {
						//check if it is reciprocal pairs
						assert(boost::out_degree(pp, paired) <= 2);
						for (boost::tie(ee, ee_end) = boost::out_edges(pp, paired); ee != ee_end; ++ee) {
							if(idx == boost::get(gsIdx, boost::target(*ee, paired))) {
								newbies.insert(pp);
								updated = true;
							}
						}
					}
				}
			}

			if(debug && newbies.size() > 0)
				verbose_message("Bin %d recruited %d contigs by paired infomation\n", good_class_ids2[k], newbies.size());

			clsV.insert(clsV.end(), newbies.begin(), newbies.end());
			clsS.insert(newbies.begin(), newbies.end());
			assert(clsV.size() == clsS.size());
		}
	}

	binned.clear();

	good_class_ids2.clear();
	good_class_ids2.insert(good_class_ids2.begin(), good_class_ids.begin(), good_class_ids.end());

	#pragma omp parallel for
	for (size_t j = 0; j < good_class_ids2.size(); ++j) {
		ContigVector& clsV = cls[good_class_ids2[j]];

		//convert to local index
		for(size_t m = 0; m < clsV.size(); ++m) {
			clsV[m] = lCtgIdx[contig_names[clsV[m]]];
			binned.insert(clsV[m]);
		}
	}

}

bool readPairFile() {
	std::ifstream is(pairFile.c_str());
	if (!is.is_open()) {
		cerr << "[Error!] can't open the paired read coverage file " << pairFile << endl;
		return false;
	}

	if(ncols(is, 1) != 3) {
		cerr << "[Error!] Number of columns in paired read coverage data file is not 3." << endl;
		return false;
	}

	paired.m_vertices.resize(seqs.size());

	int nRow = -1;
	bool isGood = true;
	size_t pastContigIdx = 0, contigIdx = 0;
	std::vector<DistancePair> contigPairs;

	for (std::string row; safeGetline(is, row) && is.good(); ++nRow) {
		if(nRow == -1) //the first row is header
			continue;

		std::stringstream ss(row);
		int c = 0;
		size_t contigIdxMate;
		double AvgCoverage;

		for (std::string col; getline(ss, col, tab_delim); ++c) {
			if (col.empty())
				break;

			if(c == 0)
				contigIdx = boost::lexical_cast<size_t>(col);
			else if(c == 1)
				contigIdxMate = boost::lexical_cast<size_t>(col);
			else if(c == 2)
				AvgCoverage = boost::lexical_cast<double>(col);
		}

		if( c != 3) {
			cerr << "[Error!] Number of columns in paired read coverage data file is not 3 in the row " << nRow + 1 << endl;
			isGood = false;
			break;
		}

		if(contigIdx >= seqs.size() || pastContigIdx >= seqs.size()) {
			cerr << "[Error!] Contig index " << contigIdx << " >= the number of total sequences " << seqs.size() << " in assembly file " << inFile << endl;
			isGood = false;
			break;
		}

		if(contigIdx == pastContigIdx) {
			DistancePair tmp(contigIdxMate, AvgCoverage);
			contigPairs.push_back(tmp);
		} else { //new index
			sort(contigPairs.begin(), contigPairs.end(), cmp_abd);

			if(contigPairs.size() == 2) {
				boost::add_edge(pastContigIdx, contigPairs[1].first, paired);
			} else if(contigPairs.size() == 3) {
				boost::add_edge(pastContigIdx, contigPairs[1].first, paired);
				boost::add_edge(pastContigIdx, contigPairs[2].first, paired);
			} else if(contigPairs.size() > 3) {
				if(contigPairs[1].second > contigPairs[3].second * minTimes)
					boost::add_edge(pastContigIdx, contigPairs[1].first, paired);
				if(contigPairs[2].second > contigPairs[3].second * minTimes)
					boost::add_edge(pastContigIdx, contigPairs[2].first, paired);
			}

			assert(boost::out_degree(pastContigIdx, paired) <= 2);

			contigPairs.clear();
			pastContigIdx = contigIdx;
		}
	}

	sort(contigPairs.begin(), contigPairs.end(), cmp_abd);

	if(contigPairs.size() == 2) {
		boost::add_edge(pastContigIdx, contigPairs[1].first, paired);
	} else if(contigPairs.size() == 3) {
		boost::add_edge(pastContigIdx, contigPairs[1].first, paired);
		boost::add_edge(pastContigIdx, contigPairs[2].first, paired);
	} else if(contigPairs.size() > 3) {
		if(contigPairs[1].second > contigPairs[3].second * minTimes)
			boost::add_edge(pastContigIdx, contigPairs[1].first, paired);
		if(contigPairs[2].second > contigPairs[3].second * minTimes)
			boost::add_edge(pastContigIdx, contigPairs[2].first, paired);
	}

	assert(boost::out_degree(contigIdx, paired) <= 2);

	if(contigIdx != seqs.size() - 1) { //the last index doesn't cover
		cerr << "[Error!] The last index does not cover all sequences given " << contigIdx << " != " << seqs.size() - 1 << endl;
		isGood = false;
	}

	if(!isGood)
		paired.clear();

	return isGood;
}

//give every members in cluster m chance to recruit other friends
void fish_more(int m, ClassMap& cls, ContigSet& leftovers) {

	ContigSet newbies;

	#pragma omp parallel for schedule (dynamic)
	for(size_t i = 0; i < cls[m].size(); ++i) {
		out_edge_iterator e, e_end;
		vertex_descriptor v = boost::vertex(cls[m][i], gprob);
		for (boost::tie(e, e_end) = boost::out_edges(v, gprob); e != e_end; ++e) {
			if(boost::get(gWgt, *e) >= p3) {
				int ff = boost::get(gIdx, boost::target(*e, gprob));
				if(leftovers.find(ff) != leftovers.end()) { //add only if it is fuzzy binning or fff is still unbinned
#pragma omp critical (FISH_MORE)
					{
						newbies.insert(ff);
//						std::cout << "new friends: " << v << " -> " << ff << " with " << boost::get(gWgt, *e) << endl;
					}
				}
			}
		}
	}

	for(ContigSet::iterator it = newbies.begin(); it != newbies.end(); ++it) {
		leftovers.erase(*it);
	}

	cls[m].insert(cls[m].end(), newbies.begin(), newbies.end());
}

void fish_more_by_friends_membership (ClassMap& cls, ContigSet& leftovers, ClassIdType& good_class_ids) {
	//profile distribution of friends and assign isolates to a bin using majority vote

	std::vector<int> clsMap(nobs, -1);

	ContigVector good_class_ids2(good_class_ids.begin(), good_class_ids.end());

	#pragma omp parallel for schedule (dynamic)
	for(size_t k = 0; k < good_class_ids2.size(); ++k) {
		for(size_t i = 0; i < cls[good_class_ids2[k]].size(); ++i) {
			assert(cls[good_class_ids2[k]][i] < (int) clsMap.size());
			clsMap[cls[good_class_ids2[k]][i]] = good_class_ids2[k];
		}
	}

	bool updated = true;

	while(updated) {
		updated = false;

		ContigVector newbies;
		ContigVector leftovers2(leftovers.begin(), leftovers.end());
		std::sort(leftovers2.begin(), leftovers2.end());

		for (size_t i = 0; i < leftovers2.size(); ++i) {
			int vid = leftovers2[i];
			out_edge_iterator e, e_end;
			vertex_descriptor v = boost::vertex(vid, gprob);

			boost::tie(e, e_end) = boost::out_edges(v, gprob);
			if(e == e_end)
				continue;

			std::unordered_map<int, int> summary;
			int _binned = 0;
			int maxFriends = 0;

			for (size_t j = 0; j < boost::out_degree(v, gprob); ++j) {
				out_edge_iterator ee = e + j;

				Similarity p = boost::get(gWgt, *ee);
				int f = boost::get(gIdx, boost::target(*ee, gprob));

				if (p >= minProb) {
					++maxFriends;
					if(clsMap[f] >= 0) { //count only binned contigs
						++_binned;
						summary[clsMap[f]]++;
					}
				}
			}

			for (std::unordered_map<int, int>::const_iterator it2 = summary.cbegin(); it2 != summary.cend(); ++it2) {
			    if (_binned > maxFriends * minBinned && it2->second > _binned/2) { //   //majority
					//cout << "Total Friends: " << maxFriends << ", Binned: " << total << ", Majority: " << it2->second << endl;
					cls[it2->first].push_back(vid);
					newbies.push_back(vid);
					clsMap[vid] = it2->first;
					updated = true;
			    	break;
			    }
			}
		}

		for(size_t i = 0; i < newbies.size(); ++i) {
			leftovers.erase(newbies[i]);
		}
	}
}

size_t fish_more_by_corr(ContigVector& medoid_ids, ClassMap& cls, ContigSet& leftovers, ClassIdType& good_class_ids) {

	double max_size = LOG10(100000);
	double min_size = LOG10(minContigByCorr);

	ContigVector leftovers2(leftovers.begin(), leftovers.end());
	std::sort(leftovers2.begin(), leftovers2.end());
	size_t fished = 0;

	ProgressTracker progress = ProgressTracker(leftovers2.size());

	#pragma omp parallel for schedule (dynamic)
	for (size_t i = 0; i < leftovers2.size(); ++i) {

		double max_corr = 0.;
		size_t which_max_corr = 0;

		for (ClassIdType::const_iterator it2 = good_class_ids.begin(); it2 != good_class_ids.end(); ++it2) {
//			if (smallCtgs.find(*it) == smallCtgs.end() && cal_tnf_dist(medoid_ids[*it2], *it) > 0.2)
//				continue;
			double corr = cal_abd_corr(medoid_ids[*it2], leftovers2[i]);
			if(corr > max_corr) { //recruiting for large bins (>=20) cls[*it2].size() >= 200; corr >= (99. - std::max(LOG10(cls[*it2].size()) - 1., 0.) * 5.)/100.
				max_corr = corr;
				which_max_corr = *it2;
			}
		}

		//1000=>90, 100000=>99
		if (max_corr >= minCorr/100.) { //smallCtgs.find(*it) == smallCtgs.end() ? minCorr * 1.05 : minCorr
			double cutCorr = ((99. - minCorr)/(max_size - min_size) * LOG10(seqs[gCtgIdx[leftovers2[i]]].size()) + (max_size*minCorr - min_size*99.)/(max_size - min_size))/100.;
			if (max_corr >= std::min(cutCorr, .99)) {
#pragma omp critical (FISH_MORE_BY_CORR)
				{
					++fished;
					cls[which_max_corr].push_back(leftovers2[i]);
				}
			}
		}

		if(!useEB) {
			progress.track();
			if (omp_get_thread_num() == 0 && progress.isStepMarker()) {
				verbose_message("fish_more_by_corr: %s\r", progress.getProgress());
			}
		}
	}

	return fished;
}

void fish_objects(int m, ContigSet& mems, Similarity p1, Similarity p2, ContigVector& medoid_ids, ContigSet& binned) { //fish (assign) objects to medoid m.
	if(debug) {
		std::cout << "---------------------" << std::endl;
		std::cout << "medoid: " << medoid_ids[m] << " with non-zero friends: " << boost::out_degree(medoid_ids[m], gprob) << std::endl;
	}

	mems.insert(medoid_ids[m]);

	out_edge_iterator e, e_end;
	vertex_descriptor v = boost::vertex(medoid_ids[m], gprob);

	int maxFriends = boost::out_degree(v, gprob);
	if(maxFriends == 0)
		return;

	boost::tie(e, e_end) = boost::out_edges(v, gprob);

	// find all friends of medoid >= p1
	#pragma omp parallel for schedule (dynamic)
	for (int i = 0; i < maxFriends; ++i) {
		out_edge_iterator ee = e + i;
		Similarity p = boost::get(gWgt, *ee);
		if(p >= p1) {
			int f = boost::get(gIdx, boost::target(*ee, gprob));
			#pragma omp critical (FISH_OBJECTS_ADD_TO_CLUSTER)
			{
				if(binned.find(f) == binned.end()) { //add only if it is fuzzy binning or f is still unbinned
					mems.insert(f);
				}
			}
		}
	}

	ContigSet newbies;

	#pragma omp parallel for schedule (dynamic)
	for(size_t i = 0; i < mems.size(); ++i) {
		ContigSet::iterator it = mems.begin();
		std::advance(it, i);
		if(*it == medoid_ids[m])
			continue;

		vertex_descriptor v = boost::vertex(*it, gprob);
		out_edge_iterator e2, e_end2;

		for (boost::tie(e2, e_end2) = boost::out_edges(v, gprob); e2 != e_end2; ++e2) {
			Similarity pp = boost::get(gWgt, *e2);
			if(pp >= p2) {
				int ff = boost::get(gIdx, boost::target(*e2, gprob));
					#pragma omp critical (FISH_OBJECTS_ADD_TO_CLUSTER_2)
					{
					if(binned.find(ff) == binned.end()) { //add only if it is fuzzy binning or ff is still unbinned
						newbies.insert(ff);
					}
					}
			}

		}
	}

	mems.insert(newbies.begin(), newbies.end());

	if(debug) {
		std::cout << "cls[m].size(): " << mems.size() << std::endl;
		for(ContigSet::iterator it = mems.begin(); it != mems.end(); ++it) {
			std::cout << *it << ", ";
		}
		std::cout << std::endl << "---------------------" << std::endl;
	}

	return;
}

void init_medoids_by_ABD(size_t k, ContigVector& medoid_ids, std::vector<double>& medoid_vals, ContigSet& binned) {
	std::list< DistancePair >::iterator it = rABD.begin();
	while (it != rABD.end()) {
		if(binned.find(it->first) == binned.end()) {
			medoid_ids[k] = it->first;
			medoid_vals[k] = it->second;
			if (debug)
				cout << "Selected medoid[" << k << "]: " << medoid_ids[k] << ", contig id: " << it->first << " with abundance " << it->second << endl;
			break; //no ++it;
		} else {
			if (useEB) {
				it->second = rand();
				rABD2.push_back(*it);
			}
			it = rABD.erase(it);
		}
	}
}

void pam_loop(int i, ContigVector& medoid_ids, std::vector<double>& medoid_vals, ContigSet& binned, ClassMap& cls) {
	init_medoids_by_ABD(i, medoid_ids, medoid_vals, binned);

	int updates = 0;
	bool updated = true;
	ContigSet medoid_prevs;
	ContigSet mems;

	while (updated) {
		updated = false;
		updates++;
		ContigSet _mems;
		fish_objects(i, _mems, p1, p2, medoid_ids, binned);

		std::vector< DistancePair > ssum(_mems.size());

		#pragma omp parallel for schedule (dynamic)
		for (size_t j = 0; j < _mems.size(); ++j) {
			ContigSet::iterator it = _mems.begin();
			std::advance(it, j);
			DistancePair s(*it, 0);
			for (ContigSet::iterator it2 = _mems.begin(); it2 != _mems.end(); ++it2) {
				if(*it != *it2)
					s.second += get_prob(*it, *it2);
			}
			ssum[j] = s;
		}

		sort(ssum.begin(), ssum.end(), cmp_abd);

		mems = _mems;

		if(medoid_prevs.find(ssum[0].first) == medoid_prevs.end()) { //preventing a loop!
			if (ssum[0].first != medoid_ids[i]) //medoid is updated
				updated = true;

			medoid_ids[i] = ssum[0].first;
			medoid_prevs.insert(medoid_ids[i]);
		}
	}

	cls[i].insert(cls[i].end(), mems.begin(), mems.end());
	binned.insert(mems.begin(), mems.end());
	if(updates > 1)
		medoid_vals[i] = std::find_if(rABD.begin(), rABD.end(), std::bind(pair_equal_to<int,double>(),medoid_ids[i]))->second;

	if (debug)
		cout << "medoid[" << i << "]: " << medoid_ids[i] << " updates: " << updates << " size: " << cls[i].size() << std::endl;
}

int pam(ContigVector& medoid_ids, std::vector<double>& medoid_vals, ContigSet& binned, ClassMap& cls, ContigSet& leftovers, ClassIdType& good_class_ids) {
	ContigVector empty;
	int goodClusters = 0;

	ProgressTracker progress(nobs - binned.size());

	while( nobs != binned.size() ) {
		medoid_ids.push_back(0);
		medoid_vals.push_back(0);

		size_t kk = medoid_ids.size() - 1;
		cls[kk] = empty;

		pam_loop(kk, medoid_ids, medoid_vals, binned, cls);

		size_t cls_size = 0;
		bool isGood = false;
		for(ContigVector::iterator it = cls[kk].begin(); it != cls[kk].end(); ++it) {
			cls_size += seqs[gCtgIdx[*it]].size();
			if(cls_size >= seedClsSize) {
				isGood = true;
				break;
			}
		}

		progress.setProgress(binned.size());
		if (!useEB && progress.isStepMarker()) {
			verbose_message("1st round binning %s\r", progress.getProgress());
		}

		if(isGood && cls[kk].size() > 2) {
			goodClusters++;
			good_class_ids.insert(kk);
		} else {
			if(cls[kk].size() > 1 || seqs[gCtgIdx[cls[kk][0]]].size() >= minContigByCorr) //keep leftovers only if it is at least valid for corr recruiting
				leftovers.insert(cls[kk].begin(), cls[kk].end());
		}
	}

	progress.setProgress(binned.size());
	if(!useEB)
		verbose_message("1st round binning %s\n", progress.getProgress());

	return goodClusters;
}

// for normal distributions
Distance cal_abd_dist2(Normal& p1, Normal& p2) {
	Distance k1, k2, tmp, d = 0;

	Distance m1 = p1.mean();
	Distance m2 = p2.mean();
	Distance v1 = p1.standard_deviation(); v1 = v1 * v1;
	Distance v2 = p2.standard_deviation(); v2 = v2 * v2;

	//normal_distribution
	if(FABS(v2 - v1) < 1e-4) {
		k1 = k2 = (m1 + m2) / 2;
	} else {
		tmp = SQRT(v1*v2 * ((m1-m2)*(m1-m2) - 2*(v1-v2)*LOG(SQRT(v2/v1))));
		k1 = (tmp - m1*v2 + m2*v1) / (v1 - v2);
		k2 = (tmp + m1*v2 - m2*v1) / (v2 - v1);
	}

	if(k1 > k2) {
		tmp = k1;
		k1 = k2;
		k2 = tmp;
	}
	if(v1 > v2) {
		std::swap(p1, p2);
	}

	if(k1 == k2)
		d += LOG(FABS(boost::math::cdf(p1, k1) - boost::math::cdf(p2, k1)));
	else
		d += LOG(FABS(boost::math::cdf(p1, k2) - boost::math::cdf(p1, k1) + boost::math::cdf(p2, k1) - boost::math::cdf(p2, k2)));

	return d;

}

// for Poisson distributions
Distance cal_abd_dist2(Poisson& p1, Poisson& p2) {
	Distance k, m1, m2;
	m1 = p1.mean();
	m2 = p2.mean();
	k = (m1 - m2) / (LOG(m1) - LOG(m2));
	return LOG(FABS(boost::math::cdf(p1, k) - boost::math::cdf(p2, k)));
}

//implementation adapted from covariance_source.c from gsl
Distance cal_abd_corr(size_t r1, size_t r2) {
	size_t i, ii;
	double sum_xsq = 0.0;
	double sum_ysq = 0.0;
	double sum_cross = 0.0;
	double ratio;
	double delta_x, delta_y;
	double mean_x = 0.0, mean_y = 0.0;
	double r = 0.0;

	size_t s = 0; //skipped

	for (i = 0; i < nABD; ++i) {
		Distance m1 = ABD(r1,i);
		Distance m2 = ABD(r2,i);

		ii = i - s;

		if(ii == 0) {
			mean_x = m1;
			mean_y = m2;
			continue;
		}

		ratio = ii / (ii + 1.0);
		delta_x = m1 - mean_x;
		delta_y = m2 - mean_y;
		sum_xsq += delta_x * delta_x * ratio;
		sum_ysq += delta_y * delta_y * ratio;
		sum_cross += delta_x * delta_y * ratio;
		mean_x += delta_x / (ii + 1.0);
		mean_y += delta_y / (ii + 1.0);
	}

	r = sum_cross / (sqrt(sum_xsq) * sqrt(sum_ysq));

	if (nABD - s < minSamples) {
		return 0;
	}

	return r;
}

Distance cal_abd_dist(size_t r1, size_t r2, int& nnz) {
	Distance d = 0;
	int nns = 0;

	assert(r1 < nobs && r2 < nobs);

	Distance m1sum = 0, m2sum = 0;
//	Distance v1sum = 0, v2sum = 0;
	for (size_t i = 0; i < nABD; ++i) {
		Distance m1 = ABD(r1,i);
		Distance m2 = ABD(r2,i);
		if (m1 > minCV || m2 > minCV) { //compare only at least one >2
			++nnz;
			m1 = std::max(m1, (Distance) 1e-6);
			m2 = std::max(m2, (Distance) 1e-6);
			if (m1 == m2) {
				++nns;
				continue;
			}

			Distance v1 = ABD_VAR(r1,i) < 1 ? 1 : ABD_VAR(r1,i);
			Distance v2 = ABD_VAR(r2,i) < 1 ? 1 : ABD_VAR(r2,i);

			Normal p1(m1, SQRT(v1)), p2(m2, SQRT(v2));
			d += cal_abd_dist2(p1, p2);
		} else {
			m1sum += m1;
			m2sum += m2;
//			v1sum += ABD_VAR(r1,i);
//			v2sum += ABD_VAR(r2,i);
		}
	}

	if (sumLowCV && (m1sum > minCV || m2sum > minCV)) {
		if (FABS(m1sum - m2sum) > 1e-3 ) {
			// now include the sum of all samples that failed the minCV test
			m1sum = std::max(m1sum, (Distance) 1e-6);
			m2sum = std::max(m2sum, (Distance) 1e-6);
			Poisson p1(m1sum), p2(m2sum);
			//Normal p1(m1sum, SQRT(v1sum)), p2(m2sum, SQRT(v2sum));
			d += cal_abd_dist2(p1, p2);
		} // else they are the same distribution, so d += 0
		++nnz;
	} else if (nnz == 0)  {
		// both samples are very low abundance, use TNF
		return 1;
	}

	if(nns == (int) nABD) //the same
		return 0;
	else
		return POW(EXP(d), 1.0 / nnz);
}

Distance cal_tnf_dist(size_t r1, size_t r2) {
	Distance d = 0;

	for (size_t i = 0; i < nTNF; ++i) {
		d += (TNF(r1,i) - TNF(r2,i)) * (TNF(r1,i) - TNF(r2,i)); //euclidean distance
	}

	d = SQRT(d);

	Distance b,c; //parameters

	size_t ctg1 = std::min(seqs[gCtgIdx[r1]].size(), (size_t)500000);
	size_t ctg2 = std::min(seqs[gCtgIdx[r2]].size(), (size_t)500000);

	Distance lw11 = LOG10(std::min(ctg1, ctg2));
	Distance lw21 = LOG10(std::max(ctg1, ctg2));
	Distance lw12 = lw11 * lw11;
	Distance lw13 = lw12 * lw11;
	Distance lw14 = lw13 * lw11;
	Distance lw15 = lw14 * lw11;
	Distance lw16 = lw15 * lw11;
	Distance lw17 = lw16 * lw11;
	Distance lw22 = lw21 * lw21;
	Distance lw23 = lw22 * lw21;
	Distance lw24 = lw23 * lw21;
	Distance lw25 = lw24 * lw21;
	Distance lw26 = lw25 * lw21;

	Distance prob;

	b = 46349.1624324381 + -76092.3748553155*lw11 + -639.918334183*lw21 + 53873.3933743949*lw12 + -156.6547554844*lw22 + -21263.6010657275*lw13 + 64.7719132839*lw23 +
			5003.2646455284*lw14 + -8.5014386744*lw24 + -700.5825500292*lw15 + 0.3968284526*lw25 + 54.037542743*lw16 + -1.7713972342*lw17 + 474.0850141891*lw11*lw21 +
			-23.966597785*lw12*lw22 + 0.7800219061*lw13*lw23 + -0.0138723693*lw14*lw24 + 0.0001027543*lw15*lw25;
	c = -443565.465710869 + 718862.10804858*lw11 + 5114.1630934534*lw21 + -501588.206183097*lw12 + 784.4442123743*lw22 + 194712.394138513*lw13 + -377.9645994741*lw23 +
			-45088.7863182741*lw14 + 50.5960513287*lw24 + 6220.3310639927*lw15 + -2.3670776453*lw25 + -473.269785487*lw16 + 15.3213264134*lw17 + -3282.8510348085*lw11*lw21 +
			164.0438603974*lw12*lw22 + -5.2778800755*lw13*lw23 + 0.0929379305*lw14*lw24 + -0.0006826817*lw15*lw25;

	//logistic model
	prob = 1.0 / ( 1 + EXP(-(b + c * d)) );

	if(prob >= .1) { //second logistic model
		b = 6770.9351457442 + -5933.7589419767*lw11 + -2976.2879986855*lw21 + 3279.7524685865*lw12 + 1602.7544794819*lw22 + -967.2906583423*lw13 + -462.0149190219*lw23 +
				159.8317289682*lw14 + 74.4884405822*lw24 + -14.0267151808*lw15 + -6.3644917671*lw25 + 0.5108811613*lw16 + 0.2252455343*lw26 + 0.965040193*lw12*lw22 +
				-0.0546309127*lw13*lw23 + 0.0012917084*lw14*lw24 + -1.14383e-05*lw15*lw25;
		c = 39406.5712626297 + -77863.1741143294*lw11 + 9586.8761567725*lw21 + 55360.1701572325*lw12 + -5825.2491611377*lw22 + -21887.8400068324*lw13 + 1751.6803621934*lw23 +
				5158.3764225203*lw14 + -290.1765894829*lw24 + -724.0348081819*lw15 + 25.364646181*lw25 + 56.0522105105*lw16 + -0.9172073892*lw26 + -1.8470088417*lw17 +
				449.4660736502*lw11*lw21 + -24.4141920625*lw12*lw22 + 0.8465834103*lw13*lw23 + -0.0158943762*lw14*lw24 + 0.0001235384*lw15*lw25;
		prob = 1.0 / ( 1 + EXP(-(b + c * d)) );
		prob = prob < .1 ? .1 : prob;
	}

	return prob;
}

//maxDist: maximum distance for further calculation (to avoid unnecessary calculation)
Distance cal_dist(size_t r1, size_t r2, Distance maxDist, bool& passed) {
	assert(smallCtgs.find(r1) == smallCtgs.end());
	assert(smallCtgs.find(r2) == smallCtgs.end());

	Distance abd_dist = 0, tnf_dist = 0;
	int nnz = 0;

	if(r1 == r2)
		return 0;

	tnf_dist = cal_tnf_dist(r1, r2);

	if(!passed && tnf_dist > maxDist) {
		return 1;
	}

	if(abdFile.length() > 0)
		abd_dist = cal_abd_dist(r1, r2, nnz);

	passed = true;

	if(tnf_dist > 0.05) { //minimum cutoff for considering abd
		return std::max(tnf_dist, abd_dist * 0.9);
	} else {
		Distance w = 0;
		if(nnz > 0)
			w = std::min(log(nnz + 1) / LOG101, 0.9); //progressive weight depending on sample sizes
		return abd_dist * w + tnf_dist * (1 - w);
	}
}

Distance cal_dist(size_t r1, size_t r2) {
	Distance maxDist = 1;
	bool passed = true;
	return cal_dist(r1, r2, maxDist, passed);
}

size_t countLines(const char* f) {
	size_t lines = 0;

	FILE * pFile;
	pFile = fopen(f, "r");

	if (pFile == NULL) {
		cerr << "[Error!] can't open input file " << f << endl;
		return 0;
	}

	while (EOF != fscanf(pFile, "%*[^\n]") && EOF != fscanf(pFile, "%*c")) {
		++lines;
	}

	fclose(pFile);

	return lines;
}

size_t ncols(std::ifstream& is, int skip = 0) {
	size_t nc = 0;

	std::string firstLine;
	while(skip-- >= 0)
		std::getline(is, firstLine);

	std::stringstream ss(firstLine);
	std::string col;
	while (std::getline(ss, col, tab_delim)) {
		++nc;
	}

	return nc;
}

size_t ncols(const char* f, int skip = 0) {
	std::ifstream is(f);
	if (!is.is_open()) {
		cerr << "[Error!] can't open input file " << f << endl;
		return 0;
	}

	return ncols(is, skip);
}

//refer to http://stackoverflow.com/questions/6089231/getting-std-ifstream-to-handle-lf-cr-and-crlf
std::istream& safeGetline(std::istream& is, std::string& t)
{
    t.clear();

    // The characters in the stream are read one-by-one using a std::streambuf.
    // That is faster than reading them one-by-one using the std::istream.
    // Code that uses streambuf this way must be guarded by a sentry object.
    // The sentry object performs various tasks,
    // such as thread synchronization and updating the stream state.

    std::istream::sentry se(is, true);
    std::streambuf* sb = is.rdbuf();

    for(;;) {
        int c = sb->sbumpc();
        switch (c) {
        case '\n':
            return is;
        case '\r':
            if(sb->sgetc() == '\n')
                sb->sbumpc();
            return is;
        case EOF:
            // Also handle the case when the last line has no line ending
            if(t.empty())
                is.setstate(std::ios::eofbit);
            return is;
        default:
            t += (char)c;
        }
    }
}
